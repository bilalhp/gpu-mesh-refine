#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <sys/time.h>

#include <main.h>

#include <cutil.h>
#include <cudpp.h>

__global__ void cu_get_counters(int *tri_edge0, int *tri_edge1, int *tri_edge2, unsigned int *tri_counter, unsigned int *tri_rev_counter, unsigned int *edge_mark_tree_next, int tri_count)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int p_counter = 0, p_rev_counter = 0;
	int e_ix;
	unsigned int mn;

	if (tid >= tri_count)
		return;

	__syncthreads();
	e_ix = tri_edge0[tid];
	mn = edge_mark_tree_next[abs(e_ix)];
	if ((__GET_E_MARK(mn) & 2)) {
		if (e_ix >= 0)
			p_counter++;
		else
			p_rev_counter++;
	}

	__syncthreads();
	e_ix = tri_edge1[tid];
	mn = edge_mark_tree_next[abs(e_ix)];
	if ((__GET_E_MARK(mn) & 2)) {
		if (e_ix >= 0)
			p_counter++;
		else
			p_rev_counter++;
	}

	__syncthreads();
	e_ix = tri_edge2[tid];
	mn = edge_mark_tree_next[abs(e_ix)];
	if ((__GET_E_MARK(mn) & 2)) {
		if (e_ix >= 0)
			p_counter++;
		else
			p_rev_counter++;
	}

	__syncthreads();
	tri_counter[tid] = p_counter;
	tri_rev_counter[tid] = p_rev_counter;
}

void cudpp_scan(void)
{
	CUDPPConfiguration config;
	config.op = CUDPP_ADD;
	config.datatype = CUDPP_UINT;
	config.algorithm = CUDPP_SCAN;
	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
	
	CUDPPHandle scanplan = 0;
	CUDPPResult result = cudppPlan(&scanplan, config, tri_count, 1, 0);
	
	if (CUDPP_SUCCESS != result) {
		printf("Error creating CUDPPPlan\n");
		exit(-1);
	}

	// Run the scan
	cudppScan(scanplan, d_tris->tri_counter_scan, d_tris->tri_counter, tri_count);
	cudppScan(scanplan, d_tris->tri_rev_counter_scan, d_tris->tri_rev_counter, tri_count);

	result = cudppDestroyPlan(scanplan);
	if (CUDPP_SUCCESS != result) {
		printf("Error destroying CUDPPPlan\n");
		exit(-1);
	}
}

void prefix_counters(void)
{
	hipError_t err;
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	int block_count = tri_count/threadsPerBlock + 1;
	dim3 dimGrid(65535, block_count/65535 + 1);

	printf(DGREEN"[%s]"NORM" WORK STARTED: #threads=%d #blocks=%d***\n", __func__, threadsPerBlock, block_count);
	fflush(stdout);
	usleep(100000);

	gettimeofday(&start_time, NULL);

	cudpp_scan();

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	cudaSafeCall(hipMemcpy(&newelemcount, &d_tris->tri_counter_scan[tri_count-1], sizeof(int), hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(&newelemcount_rev, &d_tris->tri_rev_counter_scan[tri_count-1], sizeof(int), hipMemcpyDeviceToHost));
	cudaSafeCall(hipDeviceSynchronize());

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("error!\n");
	}
}

void get_counters(void)
{
	hipError_t err;
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	int block_count = tri_count/threadsPerBlock + 1;
	dim3 dimGrid(65535, block_count/65535 + 1);

	printf(DGREEN"[%s]"NORM" WORK STARTED: #threads=%d #blocks=%d***\n", __func__, threadsPerBlock, block_count);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

#if (__CUDA_ARCH__ >= 200)
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_get_counters), hipFuncCachePreferL1);
#endif

	cu_get_counters<<<dimGrid, threadsPerBlock>>>(d_tris->tri_edge0, d_tris->tri_edge1, d_tris->tri_edge2, d_tris->tri_counter, d_tris->tri_rev_counter, d_edges->edge_mark_tree_next, tri_count);

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("error!\n");
	}
}

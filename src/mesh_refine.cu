// Includes
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>
//#include <cutil_inline.h>
#include <main.h>
#include  <signal.h>


// Variables
int debug = 0;

int threadsPerBlock = 128;
point_t *point_arr;
edge_t edges;
__device__ edge_t d_edges_s;
edge_t *d_edges = &d_edges_s;
triangle_t tris;
__device__ triangle_t d_tris_s;
triangle_t *d_tris = &d_tris_s;

unsigned int point_count, edge_count, tri_count;
unsigned int newelemcount, newelemcount_rev, new_edge_count, new_tri_count;
int refine_count;

// Host code
void usage(void)
{
	fprintf(stdout, "Usage: mesh_refine [-d] [-h] [-t <#>] [-f/i <input_file>] [-o <output_file>] [-p]\n");
	fprintf(stdout, "\t -h: print help (this output)\n");
	fprintf(stdout, "\t -d: enable debug output\n");
	fprintf(stdout, "\t -t: # of threads per block (default=256)\n");
	fprintf(stdout, "\t -f: input file\n");
	fprintf(stdout, "\t -i: input file prefix from triangle application output (input.1 for input.1.node and input.1.ele)\n");
	fprintf(stdout, "\t -p: process only on pc\n");

	exit(1);
}

unsigned long get_time_diff_us(struct timeval *start, struct timeval *end)
{
	return (end->tv_sec - start->tv_sec) * 1000000 + (end->tv_usec - start->tv_usec);
}

void sigterm(int sig)
{
	printf("Cleaning up...\n");

	device_cleanup();

	exit(0);
}

int main(int argc, char** argv)
{
	char c;
	char *input_filename = NULL;
	char *infileprefix = NULL;
	char nodefilename[128], elefilename[128];
	FILE *infile;
	char *output_filename = NULL;
	FILE *outfile;
	char output_filename_pc[1024] = {0};
	FILE *outfile_pc;
	int only_pc = 0;

	// parse arguments
	while ((c = getopt(argc, argv, "f:hdt:o:pi:")) != -1) {
		switch (c) {
		case 'h':
			usage();
		case 'd':
			debug = 1;
			break;
		case 'p':
			only_pc = 1;
			break;
		case 't':
			threadsPerBlock = atoi(optarg);
			break;
		case 'f':
			input_filename = optarg;
			break;
		case 'i':
			infileprefix = optarg;
			break;
		case 'o':
			output_filename = optarg;
			break;
		default:
			usage();
		}
	}

	dprintf("# of threads=%d infile=%s outfile=%s\n", threadsPerBlock, input_filename, output_filename);

	// input check
	if (threadsPerBlock <= 0 || (!input_filename && !infileprefix)) {
		printf("invalid arguments!\n");
		usage();
		return -1;
	}

	if (input_filename) {
		if ((infile = fopen(input_filename, "r")) == NULL)
		{
			printf("cannot open input file=%s\n", input_filename);
			return -1;
		}
	
		if (read_input_file(infile) < 0) {
			printf("error on parsing input file!\n");
			return -1;
		}
		fclose(infile);
	} else if (infileprefix) {
		sprintf(nodefilename, "%s.node", infileprefix);
		sprintf(elefilename, "%s.ele", infileprefix);

		if ((infile = fopen(nodefilename, "r")) == NULL)
		{
			printf("cannot open input file=%s\n", nodefilename);
			return -1;
		}
	
		if (read_node_file(infile) < 0) {
			printf("error on parsing node file!\n");
			return -1;
		}
		fclose(infile);

		if ((infile = fopen(elefilename, "r")) == NULL)
		{
			printf("cannot open input file=%s\n", elefilename);
			return -1;
		}
	
		if (read_ele_file(infile) < 0) {
			printf("error on parsing ele file!\n");
			return -1;
		}
		fclose(infile);
	} else {
		printf("invalid arguments ???\n");
		usage();
		return -1;
	}

	signal(SIGINT, sigterm);

	if (validate_input() < 0) {
		printf("input validation error!\n");
		return -1;
	}

#if 0
	if (debug)
		print_input();
#endif

	if (pc_alloc_copy_input()) {
		printf("cannot alloc pc input!\n");
		return -1;
	}

	if (only_pc) {
		goto process_pc;
	}

	printf("Starting process...\n");

	malloc_copy_input_to_device();

	// step 1: calculate edge lengths and mid points
	calc_edge_lengths_mid_p();

#if 1
	// bilal: workaround for floating point problem!
	cudaSafeCall(hipMemcpy(edges.edge_mid_p, d_edges->edge_mid_p, sizeof(point_t)*edge_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(edges.edge_len, d_edges->edge_len, sizeof(float)*edge_count, hipMemcpyDeviceToHost));
	pc_copy_edges();
#endif

	// step 2: mark longest edge of each triangle
	mark_longest_edges();

	// step 3: establish links
	establish_links();

	// step 4: follow links
	follow_links();

	// step 5: get the initial value of counters
	get_counters();

	// step 6: prefix the counters
	prefix_counters();

	create_new_elem_arrs();

	// step 7: refine the mesh: create new edges
	refine();

	// OKI DOKI! copy back the data to host
	copy_back();

#if 1
	if (debug) {
		copy_back();
		print_input();
	}
#endif

	// free the device stuff
	device_cleanup();

process_pc:

	if (mesh_refine_pc(only_pc)) {
		printf("MESH_REFINE_PC FAILED!\n");
	}

	if (!only_pc) {
		if (compare_results()) {
			printf(RED"VERIFICATION FAILED!\n"NORM);
		} else {
			printf("VERIFICATION COMPLETED SUCCESSFULLY!\n");
		}
	}

	if (output_filename) {
		if (!only_pc) {
			if ((outfile = fopen(output_filename, "w")) == NULL)
			{
				printf("cannot create output file=%s\n", output_filename);
				return -1;
			}
			write_output(outfile);

			fclose(outfile);
		}

		sprintf(output_filename_pc, "%s.pc", output_filename);
		if ((outfile_pc = fopen(output_filename_pc, "w")) == NULL)
		{
			printf("cannot create output file=%s\n", output_filename_pc);
			return -1;
		}

		pc_write_output(outfile_pc);

		fclose(outfile_pc);
	}

	return 0;
}

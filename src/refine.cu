#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <sys/time.h>

#include <main.h>

__global__ void cu_refine_pass3(int *new_edge0, int *new_edge1, int *orth_edge, unsigned int *edge_mark_tree_next, int *tri_edge0, int *tri_edge1, int *tri_edge2, int *tri_longest_edge, unsigned int *tri_counter_scan, unsigned int *tri_rev_counter_scan, int tri_count, int new_tri_count)
{
#define LONGEST_SELF	LONGEST
#define LONGEST_0	new_edge0[LONGEST_ABS]
#define LONGEST_1	new_edge1[LONGEST_ABS]
#define LONGEST_ORTHO	orth_edge[LONGEST_ABS]
#define RIGHT_SELF	RIGHT
#define RIGHT_0		new_edge0[RIGHT_ABS]
#define RIGHT_1		new_edge1[RIGHT_ABS]
#define RIGHT_ORTHO	orth_edge[RIGHT_ABS]
#define LEFT_SELF	LEFT
#define LEFT_0		new_edge0[LEFT_ABS]
#define LEFT_1		new_edge1[LEFT_ABS]
#define LEFT_ORTHO	orth_edge[LEFT_ABS]
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int mn;
	int LONGEST, RIGHT, LEFT;
	int e_ix0, e_ix1, e_ix2, LONGEST_ABS, RIGHT_ABS, LEFT_ABS, RIGHT_MARKED = 0, LEFT_MARKED = 0;
	int marked_count = 0;
	unsigned int created_tri_ix = tri_count - new_tri_count;
	unsigned int cnt, rcnt;

	if (tid >= (tri_count - new_tri_count))
		return;

	__syncthreads();
	cnt = tri_counter_scan[tid];
	rcnt = tri_rev_counter_scan[tid];
	created_tri_ix += __GET_T_CNT(cnt) + __GET_T_RCNT(rcnt) - 1;

	__syncthreads();
	e_ix0 = tri_edge0[tid];
	e_ix1 = tri_edge1[tid];
	e_ix2 = tri_edge2[tid];
	LONGEST = tri_longest_edge[tid];

	if (LONGEST == abs(e_ix0)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix0;
		RIGHT_ABS = abs(e_ix1);
		RIGHT = e_ix1;
		LEFT_ABS = abs(e_ix2);
		LEFT = e_ix2;
	} else if (LONGEST == abs(e_ix1)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix1;
		RIGHT_ABS = abs(e_ix2);
		RIGHT = e_ix2;
		LEFT_ABS = abs(e_ix0);
		LEFT = e_ix0;
	} else if (LONGEST == abs(e_ix2)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix2;
		RIGHT_ABS = abs(e_ix0);
		RIGHT = e_ix0;
		LEFT_ABS = abs(e_ix1);
		LEFT = e_ix1;
	} else {
		return;
	}

	__syncthreads();
	mn = edge_mark_tree_next[RIGHT_ABS];
	if ((__GET_E_MARK(mn) & 2)) {
		marked_count++;
		RIGHT_MARKED = 1;
	}
	__syncthreads();
	mn = edge_mark_tree_next[LEFT_ABS];
	if ((__GET_E_MARK(mn) & 2)) {
		marked_count++;
		LEFT_MARKED = 1;
	}

	if (LEFT_MARKED || RIGHT_MARKED) {
		// means longest is also marked
		marked_count++;
	} else {
		mn = edge_mark_tree_next[LONGEST_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			marked_count++;
		}
	}

	// START
	__syncthreads();
	if (marked_count == 1) {
/*Scenario #1*/	if (LONGEST > 0) {	// forward scenario
			// left side
			tri_edge0[tid] = -LONGEST_SELF;
			tri_edge1[tid] = LEFT_SELF;
			tri_edge2[tid] = LONGEST_0;
			// right side
			tri_edge0[created_tri_ix] = LONGEST_SELF;
			tri_edge1[created_tri_ix] = LONGEST_1;
			tri_edge2[created_tri_ix] = RIGHT_SELF;
/*Scenario #2*/	} else {	// backward scenario
			// left side
			tri_edge0[tid] = -LONGEST_ORTHO;
			tri_edge1[tid] = LEFT_SELF;
			tri_edge2[tid] = -LONGEST_1;
			// right side
			tri_edge0[created_tri_ix] = LONGEST_ORTHO;
			tri_edge1[created_tri_ix] = -LONGEST_0;
			tri_edge2[created_tri_ix] = RIGHT_SELF;
		}
	} else if (marked_count == 2) {
		// check out the right side
		if (RIGHT_MARKED) {	// right side is marked
			if (LONGEST > 0) {	// forward scenario
				// first, set the self triangle to left side
				tri_edge0[tid] = -LONGEST_SELF;
				tri_edge1[tid] = LEFT_SELF;
				tri_edge2[tid] = LONGEST_0;
				// split the right side
/* Scenario #7 */		if (RIGHT > 0) {	// forward scenario
					// left side
					tri_edge0[created_tri_ix] = -RIGHT_SELF;
					tri_edge1[created_tri_ix] = LONGEST_1;
					tri_edge2[created_tri_ix] = RIGHT_0;
					created_tri_ix--;
					// right side
					tri_edge0[created_tri_ix] = RIGHT_SELF;
					tri_edge1[created_tri_ix] = RIGHT_1;
					tri_edge2[created_tri_ix] = LONGEST_SELF;
					created_tri_ix--;
/* Scenario #9 */		} else {	// backward scenario
					// left side
					tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
					tri_edge1[created_tri_ix] = LONGEST_1;
					tri_edge2[created_tri_ix] = -RIGHT_1;
					created_tri_ix--;
					// right side
					tri_edge0[created_tri_ix] = RIGHT_ORTHO;
					tri_edge1[created_tri_ix] = -RIGHT_0;
					tri_edge2[created_tri_ix] = LONGEST_SELF;
					created_tri_ix--;
				}
			} else {	// backward scenario
				// first, set the self triangle to left side
				tri_edge0[tid] = -LONGEST_ORTHO;
				tri_edge1[tid] = LEFT_SELF;
				tri_edge2[tid] = -LONGEST_1;
				// split the right side
/* Scenario #8 */		if (RIGHT > 0) {	// forward scenario
					// left side
					tri_edge0[created_tri_ix] = -RIGHT_SELF;
					tri_edge1[created_tri_ix] = -LONGEST_0;
					tri_edge2[created_tri_ix] = RIGHT_0;
					created_tri_ix--;
					// right side
					tri_edge0[created_tri_ix] = RIGHT_SELF;
					tri_edge1[created_tri_ix] = RIGHT_1;
					tri_edge2[created_tri_ix] = LONGEST_ORTHO;
					created_tri_ix--;
/* Scenario #10 */		} else {	// backward scenario
					// left side
					tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
					tri_edge1[created_tri_ix] = -LONGEST_0;
					tri_edge2[created_tri_ix] = -RIGHT_1;
					created_tri_ix--;
					// right side
					tri_edge0[created_tri_ix] = RIGHT_ORTHO;
					tri_edge1[created_tri_ix] = -RIGHT_0;
					tri_edge2[created_tri_ix] = LONGEST_ORTHO;
					created_tri_ix--;
				}
			}
		} else {
			if (LONGEST > 0) {	// forward scenario
				// first, set the self triangle to right side
				tri_edge0[tid] = LONGEST_SELF;
				tri_edge1[tid] = LONGEST_1;
				tri_edge2[tid] = RIGHT_SELF;
				// split the left side
/* Scenario #3 */		if (LEFT > 0) {	// forward scenario
					// left side
					tri_edge0[created_tri_ix] = -LEFT_SELF;
					tri_edge1[created_tri_ix] = -LONGEST_SELF;
					tri_edge2[created_tri_ix] = LEFT_0;
					created_tri_ix--;
					// right side
					tri_edge0[created_tri_ix] = LEFT_SELF;
					tri_edge1[created_tri_ix] = LEFT_1;
					tri_edge2[created_tri_ix] = LONGEST_0;
					created_tri_ix--;
/* Scenario #5 */		} else {	// backward scenario
					// left side
					tri_edge0[created_tri_ix] = -LEFT_ORTHO;
					tri_edge1[created_tri_ix] = -LONGEST_SELF;
					tri_edge2[created_tri_ix] = -LEFT_1;
					created_tri_ix--;
					// right side
					tri_edge0[created_tri_ix] = LEFT_ORTHO;
					tri_edge1[created_tri_ix] = -LEFT_0;
					tri_edge2[created_tri_ix] = LONGEST_0;
					created_tri_ix--;
				}
			} else {	// backward scenario
				// first, set the self triangle to right side
				tri_edge0[tid] = LONGEST_ORTHO;
				tri_edge1[tid] = -LONGEST_0;
				tri_edge2[tid] = RIGHT_SELF;
				// split the left side
/* Scenario #4 */		if (LEFT > 0) {	// forward scenario
					// left side
					tri_edge0[created_tri_ix] = -LEFT_SELF;
					tri_edge1[created_tri_ix] = -LONGEST_ORTHO;
					tri_edge2[created_tri_ix] = LEFT_0;
					created_tri_ix--;
					// right side
					tri_edge0[created_tri_ix] = LEFT_SELF;
					tri_edge1[created_tri_ix] = LEFT_1;
					tri_edge2[created_tri_ix] = -LONGEST_1;
					created_tri_ix--;
/* Scenario #6 */		} else {	// backward scenario
					// left side
					tri_edge0[created_tri_ix] = -LEFT_ORTHO;
					tri_edge1[created_tri_ix] = -LONGEST_ORTHO;
					tri_edge2[created_tri_ix] = -LEFT_1;
					created_tri_ix--;
					// right side
					tri_edge0[created_tri_ix] = LEFT_ORTHO;
					tri_edge1[created_tri_ix] = -LEFT_0;
					tri_edge2[created_tri_ix] = -LONGEST_1;
					created_tri_ix--;
				}
			}
		}
	} else if (marked_count == 3) {
		if (LONGEST > 0) {	// longest forward scenario
			if (LEFT > 0) {	// left forward scenario
/* Scenario #11 */		if (RIGHT > 0) {	// right forward scenario
					// first, set the self triangle to part 1
					tri_edge0[tid] = -LONGEST_SELF;
					tri_edge1[tid] = LEFT_0;
					tri_edge2[tid] = -LEFT_SELF;
					// part 2
					tri_edge0[created_tri_ix] = LEFT_SELF;
					tri_edge1[created_tri_ix] = LEFT_1;
					tri_edge2[created_tri_ix] = LONGEST_0;
					created_tri_ix--;
					// part 3
					tri_edge0[created_tri_ix] = -RIGHT_SELF;
					tri_edge1[created_tri_ix] = LONGEST_1;
					tri_edge2[created_tri_ix] = RIGHT_0;
					created_tri_ix--;
					// part 4
					tri_edge0[created_tri_ix] = LONGEST_SELF;
					tri_edge1[created_tri_ix] = RIGHT_SELF;
					tri_edge2[created_tri_ix] = RIGHT_1;
					created_tri_ix--;
/* Scenario #12 */		} else {	// right backward scenario
					// first, set the self triangle to part 1
					tri_edge0[tid] = -LONGEST_SELF;
					tri_edge1[tid] = LEFT_0;
					tri_edge2[tid] = -LEFT_SELF;
					// part 2
					tri_edge0[created_tri_ix] = LEFT_SELF;
					tri_edge1[created_tri_ix] = LEFT_1;
					tri_edge2[created_tri_ix] = LONGEST_0;
					created_tri_ix--;
					// part 3
					tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
					tri_edge1[created_tri_ix] = LONGEST_1;
					tri_edge2[created_tri_ix] = -RIGHT_1;
					created_tri_ix--;
					// part 4
					tri_edge0[created_tri_ix] = LONGEST_SELF;
					tri_edge1[created_tri_ix] = RIGHT_ORTHO;
					tri_edge2[created_tri_ix] = -RIGHT_0;
					created_tri_ix--;
				}
			} else {	// left backward scenario
/* Scenario #14 */		if (RIGHT > 0) {	// right forward scenario
					// first, set the self triangle to part 1
					tri_edge0[tid] = -LONGEST_SELF;
					tri_edge1[tid] = -LEFT_1;
					tri_edge2[tid] = -LEFT_ORTHO;
					// part 2
					tri_edge0[created_tri_ix] = LEFT_ORTHO;
					tri_edge1[created_tri_ix] = -LEFT_0;
					tri_edge2[created_tri_ix] = LONGEST_0;
					created_tri_ix--;
					// part 3
					tri_edge0[created_tri_ix] = -RIGHT_SELF;
					tri_edge1[created_tri_ix] = LONGEST_1;
					tri_edge2[created_tri_ix] = RIGHT_0;
					created_tri_ix--;
					// part 4
					tri_edge0[created_tri_ix] = LONGEST_SELF;
					tri_edge1[created_tri_ix] = RIGHT_SELF;
					tri_edge2[created_tri_ix] = RIGHT_1;
					created_tri_ix--;
/* Scenario #13 */		} else {	// right backward scenario
					// first, set the self triangle to part 1
					tri_edge0[tid] = -LONGEST_SELF;
					tri_edge1[tid] = -LEFT_1;
					tri_edge2[tid] = -LEFT_ORTHO;
					// part 2
					tri_edge0[created_tri_ix] = LEFT_ORTHO;
					tri_edge1[created_tri_ix] = -LEFT_0;
					tri_edge2[created_tri_ix] = LONGEST_0;
					created_tri_ix--;
					// part 3
					tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
					tri_edge1[created_tri_ix] = LONGEST_1;
					tri_edge2[created_tri_ix] = -RIGHT_1;
					created_tri_ix--;
					// part 4
					tri_edge0[created_tri_ix] = LONGEST_SELF;
					tri_edge1[created_tri_ix] = RIGHT_ORTHO;
					tri_edge2[created_tri_ix] = -RIGHT_0;
					created_tri_ix--;
				}
			}
		} else {	// longest backward scenario
			if (LEFT > 0) {	// left forward scenario
/* Scenario #15 */		if (RIGHT > 0) {	// right forward scenario
					// first, set the self triangle to part 1
					tri_edge0[tid] = -LONGEST_ORTHO;
					tri_edge1[tid] = LEFT_0;
					tri_edge2[tid] = -LEFT_SELF;
					// part 2
					tri_edge0[created_tri_ix] = LEFT_SELF;
					tri_edge1[created_tri_ix] = LEFT_1;
					tri_edge2[created_tri_ix] = -LONGEST_1;
					created_tri_ix--;
					// part 3
					tri_edge0[created_tri_ix] = -RIGHT_SELF;
					tri_edge1[created_tri_ix] = -LONGEST_0;
					tri_edge2[created_tri_ix] = RIGHT_0;
					created_tri_ix--;
					// part 4
					tri_edge0[created_tri_ix] = LONGEST_ORTHO;
					tri_edge1[created_tri_ix] = RIGHT_SELF;
					tri_edge2[created_tri_ix] = RIGHT_1;
					created_tri_ix--;
/* Scenario #16 */		} else {	// right backward scenario
					// first, set the self triangle to part 1
					tri_edge0[tid] = -LONGEST_ORTHO;
					tri_edge1[tid] = LEFT_0;
					tri_edge2[tid] = -LEFT_SELF;
					// part 2
					tri_edge0[created_tri_ix] = LEFT_SELF;
					tri_edge1[created_tri_ix] = LEFT_1;
					tri_edge2[created_tri_ix] = -LONGEST_1;
					created_tri_ix--;
					// part 3
					tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
					tri_edge1[created_tri_ix] = -LONGEST_0;
					tri_edge2[created_tri_ix] = -RIGHT_1;
					created_tri_ix--;
					// part 4
					tri_edge0[created_tri_ix] = LONGEST_ORTHO;
					tri_edge1[created_tri_ix] = RIGHT_ORTHO;
					tri_edge2[created_tri_ix] = -RIGHT_0;
					created_tri_ix--;
				}
			} else {	// left backward scenario
/* Scenario #18 */		if (RIGHT > 0) {	// right forward scenario
					// first, set the self triangle to part 1
					tri_edge0[tid] = -LONGEST_ORTHO;
					tri_edge1[tid] = -LEFT_1;
					tri_edge2[tid] = -LEFT_ORTHO;
					// part 2
					tri_edge0[created_tri_ix] = LEFT_ORTHO;
					tri_edge1[created_tri_ix] = -LEFT_0;
					tri_edge2[created_tri_ix] = -LONGEST_1;
					created_tri_ix--;
					// part 3
					tri_edge0[created_tri_ix] = -RIGHT_SELF;
					tri_edge1[created_tri_ix] = -LONGEST_0;
					tri_edge2[created_tri_ix] = RIGHT_0;
					created_tri_ix--;
					// part 4
					tri_edge0[created_tri_ix] = LONGEST_ORTHO;
					tri_edge1[created_tri_ix] = RIGHT_SELF;
					tri_edge2[created_tri_ix] = RIGHT_1;
					created_tri_ix--;
/* Scenario #17 */		} else {	// right backward scenario
					// first, set the self triangle to part 1
					tri_edge0[tid] = -LONGEST_ORTHO;
					tri_edge1[tid] = -LEFT_1;
					tri_edge2[tid] = -LEFT_ORTHO;
					// part 2
					tri_edge0[created_tri_ix] = LEFT_ORTHO;
					tri_edge1[created_tri_ix] = -LEFT_0;
					tri_edge2[created_tri_ix] = -LONGEST_1;
					created_tri_ix--;
					// part 3
					tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
					tri_edge1[created_tri_ix] = -LONGEST_0;
					tri_edge2[created_tri_ix] = -RIGHT_1;
					created_tri_ix--;
					// part 4
					tri_edge0[created_tri_ix] = LONGEST_ORTHO;
					tri_edge1[created_tri_ix] = RIGHT_ORTHO;
					tri_edge2[created_tri_ix] = -RIGHT_0;
					created_tri_ix--;
				}
			}
		}
	}
}

static __device__ point_t get_opposite_point(int i, point_t *edge_point0, point_t *edge_point1, int *new_edge0, int *new_edge1)
{
	int i_abs = abs(i);
	int ne0, ne1;

	ne0 = new_edge0[i_abs];
	if (ne0 > 0) {	// is it divided?
		if (i > 0) {	// forward
			ne1 = new_edge1[i_abs];
			return edge_point1[abs(ne1)];
		} else {
			return edge_point0[abs(ne0)];
		}
	} else {
		if (i > 0) {
			return edge_point1[i_abs];
		} else {
			return edge_point0[i_abs];
		}
	}
}

__global__ void cu_refine_pass2(point_t *edge_point0, point_t *edge_point1, int *new_edge0, int *new_edge1, point_t *edge_mid_p, unsigned int *edge_mark_tree_next, int *tri_edge0, int *tri_edge1, int *tri_edge2, int *tri_longest_edge, int tri_count, int new_tri_count)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int mn;
	int LONGEST, RIGHT, LEFT;
	int e_ix0, e_ix1, e_ix2, LONGEST_ABS, RIGHT_ABS, LEFT_ABS;

	if (tid >= (tri_count - new_tri_count))
		return;

	__syncthreads();
	e_ix0 = tri_edge0[tid];
	e_ix1 = tri_edge1[tid];
	e_ix2 = tri_edge2[tid];
	LONGEST = tri_longest_edge[tid];

	if (LONGEST == abs(e_ix0)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix0;
		RIGHT_ABS = abs(e_ix1);
		RIGHT = e_ix1;
		LEFT_ABS = abs(e_ix2);
		LEFT = e_ix2;
	} else if (LONGEST == abs(e_ix1)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix1;
		RIGHT_ABS = abs(e_ix2);
		RIGHT = e_ix2;
		LEFT_ABS = abs(e_ix0);
		LEFT = e_ix0;
	} else if (LONGEST == abs(e_ix2)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix2;
		RIGHT_ABS = abs(e_ix0);
		RIGHT = e_ix0;
		LEFT_ABS = abs(e_ix1);
		LEFT = e_ix1;
	}

	// process longest
	__syncthreads();
	mn = edge_mark_tree_next[LONGEST_ABS];
	if ((__GET_E_MARK(mn) & 2)) {
		if (LONGEST > 0) {	// forward scenario
			edge_point0[LONGEST_ABS] = get_opposite_point(RIGHT, edge_point0, edge_point1, new_edge0, new_edge1);
			edge_point1[LONGEST_ABS] = edge_mid_p[LONGEST_ABS];
		}
	}

	// process right
	__syncthreads();
	mn = edge_mark_tree_next[RIGHT_ABS];
	if ((__GET_E_MARK(mn) & 2)) {
		if (RIGHT > 0) {	// forward scenario
			edge_point0[RIGHT_ABS] = edge_mid_p[LONGEST_ABS];
			edge_point1[RIGHT_ABS] = edge_mid_p[RIGHT_ABS];
		}
	}

	// process left
	__syncthreads();
	mn = edge_mark_tree_next[LEFT_ABS];
	if ((__GET_E_MARK(mn) & 2)) {
		if (LEFT > 0) {	// forward scenario
			edge_point0[LEFT_ABS] = edge_mid_p[LONGEST_ABS];
			edge_point1[LEFT_ABS] = edge_mid_p[LEFT_ABS];
		}
	}
}

__global__ void cu_refine_pass1(point_t *edge_point0, point_t *edge_point1, point_t *edge_mid_p, unsigned int *edge_mark_tree_next, int *new_edge0, int *new_edge1, int *orth_edge, int *tri_edge0, int *tri_edge1, int *tri_edge2, int *tri_longest_edge, unsigned int *tri_counter_scan, unsigned int *tri_rev_counter_scan, int edge_count, int tri_count, int new_edge_count, int new_tri_count)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int created_edge_ix = edge_count - new_edge_count;
	unsigned int mn;
	int LONGEST, RIGHT, LEFT;
	int e_ix0, e_ix1, e_ix2, LONGEST_ABS, RIGHT_ABS, LEFT_ABS;
	unsigned int cnt, rcnt;

	if (tid >= (tri_count - new_tri_count))
		return;

	__syncthreads();
	cnt = tri_counter_scan[tid];
	rcnt = tri_rev_counter_scan[tid];
	created_edge_ix += 2*__GET_T_CNT(cnt) + __GET_T_RCNT(rcnt) - 1;
	e_ix0 = tri_edge0[tid];
	e_ix1 = tri_edge1[tid];
	e_ix2 = tri_edge2[tid];
	LONGEST = tri_longest_edge[tid];

	if (LONGEST == abs(e_ix0)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix0;
		RIGHT_ABS = abs(e_ix1);
		RIGHT = e_ix1;
		LEFT_ABS = abs(e_ix2);
		LEFT = e_ix2;
	} else if (LONGEST == abs(e_ix1)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix1;
		RIGHT_ABS = abs(e_ix2);
		RIGHT = e_ix2;
		LEFT_ABS = abs(e_ix0);
		LEFT = e_ix0;
	} else if (LONGEST == abs(e_ix2)) {
		LONGEST_ABS = LONGEST;
		LONGEST = e_ix2;
		RIGHT_ABS = abs(e_ix0);
		RIGHT = e_ix0;
		LEFT_ABS = abs(e_ix1);
		LEFT = e_ix1;
	}

	// process longest
	__syncthreads();
	mn = edge_mark_tree_next[LONGEST_ABS];
	if ((__GET_E_MARK(mn) & 2)) {
		if (LONGEST > 0) {	// forward scenario
			edge_point0[created_edge_ix] = edge_point0[LONGEST_ABS];
			edge_point1[created_edge_ix] = edge_mid_p[LONGEST_ABS];
			new_edge0[LONGEST_ABS] = created_edge_ix;
			created_edge_ix--;
			edge_point0[created_edge_ix] = edge_mid_p[LONGEST_ABS];
			edge_point1[created_edge_ix] = edge_point1[LONGEST_ABS];
			new_edge1[LONGEST_ABS] = created_edge_ix;
			created_edge_ix--;
		} else {
			edge_point0[created_edge_ix] = (RIGHT > 0) ? edge_point1[RIGHT_ABS] : edge_point0[RIGHT_ABS];
			edge_point1[created_edge_ix] = edge_mid_p[LONGEST_ABS];
			orth_edge[LONGEST_ABS] = created_edge_ix;
			created_edge_ix--;
		}
	}

	// process right
	__syncthreads();
	mn = edge_mark_tree_next[RIGHT_ABS];
	if ((__GET_E_MARK(mn) & 2)) {
		if (RIGHT > 0) {	// forward scenario
			edge_point0[created_edge_ix] = edge_point0[RIGHT_ABS];
			edge_point1[created_edge_ix] = edge_mid_p[RIGHT_ABS];
			new_edge0[RIGHT_ABS] = created_edge_ix;
			created_edge_ix--;
			edge_point0[created_edge_ix] = edge_mid_p[RIGHT_ABS];
			edge_point1[created_edge_ix] = edge_point1[RIGHT_ABS];
			new_edge1[RIGHT_ABS] = created_edge_ix;
			created_edge_ix--;
		} else {
			edge_point0[created_edge_ix] = edge_mid_p[LONGEST_ABS];
			edge_point1[created_edge_ix] = edge_mid_p[RIGHT_ABS];
			orth_edge[RIGHT_ABS] = created_edge_ix;
			created_edge_ix--;
		}
	}

	// process left
	__syncthreads();
	mn = edge_mark_tree_next[LEFT_ABS];
	if ((__GET_E_MARK(mn) & 2)) {
		if (LEFT > 0) {	// forward scenario
			edge_point0[created_edge_ix] = edge_point0[LEFT_ABS];
			edge_point1[created_edge_ix] = edge_mid_p[LEFT_ABS];
			new_edge0[LEFT_ABS] = created_edge_ix;
			created_edge_ix--;
			edge_point0[created_edge_ix] = edge_mid_p[LEFT_ABS];
			edge_point1[created_edge_ix] = edge_point1[LEFT_ABS];
			new_edge1[LEFT_ABS] = created_edge_ix;
			created_edge_ix--;
		} else {
			edge_point0[created_edge_ix] = edge_mid_p[LONGEST_ABS];
			edge_point1[created_edge_ix] = edge_mid_p[LEFT_ABS];
			orth_edge[LEFT_ABS] = created_edge_ix;
			created_edge_ix--;
		}
	}
}

void refine(void)
{
	hipError_t err;
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	int block_count = (tri_count- new_tri_count)/threadsPerBlock + 1;
	dim3 dimGrid(65535, block_count/65535 + 1);

	printf(DGREEN"[%s]"NORM" WORK STARTED: #threads=%d #blocks=%d***\n", __func__, threadsPerBlock, block_count);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

#if (__CUDA_ARCH__ >= 200)
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_refine_pass1), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_refine_pass2), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_refine_pass3), hipFuncCachePreferL1);
#endif

	cu_refine_pass1<<<dimGrid, threadsPerBlock>>>(d_edges->edge_point0, d_edges->edge_point1, d_edges->edge_mid_p, d_edges->edge_mark_tree_next, d_edges->new_edge0, d_edges->new_edge1, d_edges->orth_edge, d_tris->tri_edge0, d_tris->tri_edge1, d_tris->tri_edge2, d_tris->tri_longest_edge, d_tris->tri_counter_scan, d_tris->tri_rev_counter_scan, edge_count, tri_count, new_edge_count, new_tri_count);

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" PASS1 TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("error!\n");
	}

	cu_refine_pass2<<<dimGrid, threadsPerBlock>>>(d_edges->edge_point0, d_edges->edge_point1, d_edges->new_edge0, d_edges->new_edge1, d_edges->edge_mid_p, d_edges->edge_mark_tree_next, d_tris->tri_edge0, d_tris->tri_edge1, d_tris->tri_edge2, d_tris->tri_longest_edge, tri_count, new_tri_count);

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" PASS2 TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {       
		printf("error!\n");
	}

	cu_refine_pass3<<<dimGrid, threadsPerBlock>>>(d_edges->new_edge0, d_edges->new_edge1, d_edges->orth_edge, d_edges->edge_mark_tree_next, d_tris->tri_edge0, d_tris->tri_edge1, d_tris->tri_edge2, d_tris->tri_longest_edge, d_tris->tri_counter_scan, d_tris->tri_rev_counter_scan, tri_count, new_tri_count);

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" PASS3 TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("error!\n");
	}
}

int host_realloc_tris(void)
{
	tris.tri_edge0 = (int *)realloc(tris.tri_edge0, (tri_count + new_tri_count)*sizeof(int));
	if (tris.tri_edge0 == NULL) {
		printf("cannot realloc tris.tri_edge0!\n");
		return -1;
	}

	tris.tri_edge1 = (int *)realloc(tris.tri_edge1, (tri_count + new_tri_count)*sizeof(int));
	if (tris.tri_edge1 == NULL) {
		printf("cannot realloc tris.tri_edge1!\n");
		return -1;
	}

	tris.tri_edge2 = (int *)realloc(tris.tri_edge2, (tri_count + new_tri_count)*sizeof(int));
	if (tris.tri_edge2 == NULL) {
		printf("cannot realloc tris.tri_edge2!\n");
		return -1;
	}

	tris.tri_longest_edge = (int *)realloc(tris.tri_longest_edge, (tri_count + new_tri_count)*sizeof(int));
	if (tris.tri_longest_edge == NULL) {
		printf("cannot realloc tris.tri_longest_edge!\n");
		return -1;
	}

	tris.tri_counter = (unsigned int *)realloc(tris.tri_counter, (tri_count + new_tri_count)*sizeof(int));
	if (tris.tri_counter == NULL) {
		printf("cannot realloc tris.tri_counter!\n");
		return -1;
	}

	tris.tri_counter_scan = (unsigned int *)realloc(tris.tri_counter_scan, (tri_count + new_tri_count)*sizeof(int));
	if (tris.tri_counter_scan == NULL) {
		printf("cannot realloc tris.tri_counter_scan!\n");
		return -1;
	}

	tris.tri_rev_counter = (unsigned int *)realloc(tris.tri_rev_counter, (tri_count + new_tri_count)*sizeof(int));
	if (tris.tri_rev_counter == NULL) {
		printf("cannot realloc tris.tri_rev_counter!\n");
		return -1;
	}

	tris.tri_rev_counter_scan = (unsigned int *)realloc(tris.tri_rev_counter_scan, (tri_count + new_tri_count)*sizeof(int));
	if (tris.tri_rev_counter_scan == NULL) {
		printf("cannot realloc tris.tri_rev_counter_scan!\n");
		return -1;
	}

	return 0;
}

int host_realloc_edges(void)
{
	edges.edge_point0 = (point_t *)realloc(edges.edge_point0, (edge_count + new_edge_count)*sizeof(point_t));
	if (edges.edge_point0 == NULL) {
		printf("cannot realloc edge_point0 arr!\n");
		return -1;
	}

	edges.edge_point1 = (point_t *)realloc(edges.edge_point1, (edge_count + new_edge_count)*sizeof(point_t));
	if (edges.edge_point1 == NULL) {
		printf("cannot realloc edge_point1 arr!\n");
		return -1;
	}

	edges.edge_mid_p = (point_t *)realloc(edges.edge_mid_p, (edge_count + new_edge_count)*sizeof(point_t));
	if (edges.edge_mid_p == NULL) {
		printf("cannot realloc edge_mid_p arr!\n");
		return -1;
	}

	edges.edge_len = (float *)realloc(edges.edge_len, (edge_count + new_edge_count)*sizeof(float));
	if (edges.edge_len == NULL) {
		printf("cannot realloc edge_len arr!\n");
		return -1;
	}

	edges.edge_mark_tree_next = (unsigned int *)realloc(edges.edge_mark_tree_next, (edge_count + new_edge_count)*sizeof(int));
	if (edges.edge_mark_tree_next == NULL) {
		printf("cannot realloc edge_mark_tree_next arr!\n");
		return -1;
	}

	edges.new_edge0 = (int *)realloc(edges.new_edge0, (edge_count + new_edge_count)*sizeof(int));
	if (edges.new_edge0 == NULL) {
		printf("cannot realloc new_edge0 arr!\n");
		return -1;
	}

	edges.new_edge1 = (int *)realloc(edges.new_edge1, (edge_count + new_edge_count)*sizeof(int));
	if (edges.new_edge1 == NULL) {
		printf("cannot realloc new_edge1 arr!\n");
		return -1;
	}

	edges.orth_edge = (int *)realloc(edges.orth_edge, (edge_count + new_edge_count)*sizeof(int));
	if (edges.orth_edge == NULL) {
		printf("cannot realloc orth_edge arr!\n");
		return -1;
	}

	return 0;
}

void create_new_elem_arrs(void)
{
	new_edge_count = 2*__GET_T_CNT(newelemcount) + __GET_T_RCNT(newelemcount_rev);
	new_tri_count = __GET_T_CNT(newelemcount) + __GET_T_RCNT(newelemcount_rev);

	printf("new_edge_count=%d new_tri_count=%d\n", new_edge_count, new_tri_count);

#if 0
	if ((__GET_T_CNT(newelemcount) < refine_count) || (__GET_T_RCNT(newelemcount) < refine_count)) {
		printf(RED"FAILED!!\n"NORM);
		device_cleanup();
		exit(1);
	}
#endif
	// copy back the modified data from device
	copy_back();

	// free the old device stuff
	device_cleanup();

	if (host_realloc_edges() < 0) {
		printf("host realloc error\n");
		exit(1);
	}

	if (host_realloc_tris() < 0) {
		printf("host realloc error\n");
		exit(1);
	}

	edge_count += new_edge_count;
	tri_count += new_tri_count;

	// malloc and copy the new arrays
	malloc_copy_input_to_device2();
}

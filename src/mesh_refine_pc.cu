#include "hip/hip_runtime.h"
// Includes
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>
//#include <cutil_inline.h>
#include <main.h>


#define PC_E_P0(_i)		pc_edges.edge_point0[_i]
#define PC_E_P1(_i)		pc_edges.edge_point1[_i]
#define PC_E_MIDP(_i)		pc_edges.edge_mid_p[_i]
#define PC_E_MN(_i)		pc_edges.edge_mark_tree_next[_i]
#define PC_E_NE0(_i)		pc_edges.new_edge0[_i]
#define PC_E_NE1(_i)		pc_edges.new_edge1[_i]
#define PC_E_OE(_i)		pc_edges.orth_edge[_i]
#define PC_E_LEN(_i)		pc_edges.edge_len[_i]
#define PC_T_E0(_i)		pc_tris.tri_edge0[_i]
#define PC_T_E1(_i)		pc_tris.tri_edge1[_i]
#define PC_T_E2(_i)		pc_tris.tri_edge2[_i]
#define PC_T_LE(_i)		pc_tris.tri_longest_edge[_i]
#define PC_T_CNT(_i)		pc_tris.tri_counter[_i]
#define PC_T_CNT_SCAN(_i)	pc_tris.tri_counter_scan[_i]
#define PC_T_RCNT(_i)		pc_tris.tri_rev_counter[_i]
#define PC_T_RCNT_SCAN(_i)	pc_tris.tri_rev_counter_scan[_i]

#define PC_FOR_EACH_T	for (int i = 0; i < pc_tri_count; i++)
#define PC_FOR_EACH_E	for (int i = 1; i < pc_edge_count; i++)
#define PC_FOR_EACH_P	for (int i = 0; i < pc_point_count; i++)

#define PC_GET_E_MARK(_i)		__GET_E_MARK(PC_E_MN(i))
#define PC_GET_E_NEXT(_i)		__GET_E_NEXT(PC_E_MN(i))

#define PC_GET_T_CNT(_i)		__GET_T_CNT(PC_T_CNT(_i))
#define PC_GET_T_CNT_SCAN(_i)		__GET_T_CNT(PC_T_CNT_SCAN(_i))
#define PC_GET_T_RCNT(_i)		__GET_T_RCNT(PC_T_RCNT(_i))
#define PC_GET_T_RCNT_SCAN(_i)		__GET_T_RCNT(PC_T_RCNT_SCAN(_i))


edge_t pc_edges;
triangle_t pc_tris;

unsigned int pc_edge_count, pc_tri_count;
unsigned int pc_newelemcount, pc_newelemcount_rev, pc_new_edge_count, pc_new_tri_count;


void refine_pass3(void)
{
	unsigned int *edge_mark_tree_next = pc_edges.edge_mark_tree_next;
	int *new_edge0 = pc_edges.new_edge0;
	int *new_edge1 = pc_edges.new_edge1;
	int *orth_edge = pc_edges.orth_edge;
	int *tri_edge0 = pc_tris.tri_edge0;
	int *tri_edge1 = pc_tris.tri_edge1;
	int *tri_edge2 = pc_tris.tri_edge2;
	int *tri_longest_edge = pc_tris.tri_longest_edge;
	unsigned int *tri_counter_scan = pc_tris.tri_counter_scan;
	unsigned int *tri_rev_counter_scan = pc_tris.tri_rev_counter_scan;

	PC_FOR_EACH_T {
#define LONGEST_SELF	LONGEST
#define LONGEST_0	new_edge0[LONGEST_ABS]
#define LONGEST_1	new_edge1[LONGEST_ABS]
#define LONGEST_ORTHO	orth_edge[LONGEST_ABS]
#define RIGHT_SELF	RIGHT
#define RIGHT_0		new_edge0[RIGHT_ABS]
#define RIGHT_1		new_edge1[RIGHT_ABS]
#define RIGHT_ORTHO	orth_edge[RIGHT_ABS]
#define LEFT_SELF	LEFT
#define LEFT_0		new_edge0[LEFT_ABS]
#define LEFT_1		new_edge1[LEFT_ABS]
#define LEFT_ORTHO	orth_edge[LEFT_ABS]
		unsigned int tid = i;
		unsigned int mn;
		int LONGEST, RIGHT, LEFT;
		int e_ix0, e_ix1, e_ix2, LONGEST_ABS, RIGHT_ABS, LEFT_ABS, RIGHT_MARKED = 0, LEFT_MARKED = 0;
		int marked_count = 0;
		unsigned int created_tri_ix = tri_count - new_tri_count;
		unsigned int cnt, rcnt;

		if (tid >= (pc_tri_count - pc_new_tri_count))
			return;

		cnt = tri_counter_scan[tid];
		rcnt = tri_rev_counter_scan[tid];
		created_tri_ix += __GET_T_CNT(cnt) + __GET_T_RCNT(rcnt) - 1;

		e_ix0 = tri_edge0[tid];
		e_ix1 = tri_edge1[tid];
		e_ix2 = tri_edge2[tid];
		LONGEST = tri_longest_edge[tid];

		if (LONGEST == abs(e_ix0)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix0;
			RIGHT_ABS = abs(e_ix1);
			RIGHT = e_ix1;
			LEFT_ABS = abs(e_ix2);
			LEFT = e_ix2;
		} else if (LONGEST == abs(e_ix1)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix1;
			RIGHT_ABS = abs(e_ix2);
			RIGHT = e_ix2;
			LEFT_ABS = abs(e_ix0);
			LEFT = e_ix0;
		} else if (LONGEST == abs(e_ix2)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix2;
			RIGHT_ABS = abs(e_ix0);
			RIGHT = e_ix0;
			LEFT_ABS = abs(e_ix1);
			LEFT = e_ix1;
		} else {
			return;
		}

		mn = edge_mark_tree_next[RIGHT_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			marked_count++;
			RIGHT_MARKED = 1;
		}
		mn = edge_mark_tree_next[LEFT_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			marked_count++;
			LEFT_MARKED = 1;
		}

		if (LEFT_MARKED || RIGHT_MARKED) {
			// means longest is also marked
			marked_count++;
		} else {
			mn = edge_mark_tree_next[LONGEST_ABS];
			if ((__GET_E_MARK(mn) & 2)) {
				marked_count++;
			}
		}

		// START
		if (marked_count == 1) {
	/*Scenario #1*/	if (LONGEST > 0) {	// forward scenario
				dprintf("scenario 1\n");
				// left side
				tri_edge0[tid] = -LONGEST_SELF;
				tri_edge1[tid] = LEFT_SELF;
				tri_edge2[tid] = LONGEST_0;
				// right side
				tri_edge0[created_tri_ix] = LONGEST_SELF;
				tri_edge1[created_tri_ix] = LONGEST_1;
				tri_edge2[created_tri_ix] = RIGHT_SELF;
	/*Scenario #2*/	} else {	// backward scenario
				dprintf("scenario 2\n");
				// left side
				tri_edge0[tid] = -LONGEST_ORTHO;
				tri_edge1[tid] = LEFT_SELF;
				tri_edge2[tid] = -LONGEST_1;
				// right side
				tri_edge0[created_tri_ix] = LONGEST_ORTHO;
				tri_edge1[created_tri_ix] = -LONGEST_0;
				tri_edge2[created_tri_ix] = RIGHT_SELF;
			}
		} else if (marked_count == 2) {
			// check out the right side
			if (RIGHT_MARKED) {	// right side is marked
				if (LONGEST > 0) {	// forward scenario
					// first, set the self triangle to left side
					tri_edge0[tid] = -LONGEST_SELF;
					tri_edge1[tid] = LEFT_SELF;
					tri_edge2[tid] = LONGEST_0;
					// split the right side
	/* Scenario #7 */		if (RIGHT > 0) {	// forward scenario
						dprintf("scenario 7\n");
						// left side
						tri_edge0[created_tri_ix] = -RIGHT_SELF;
						tri_edge1[created_tri_ix] = LONGEST_1;
						tri_edge2[created_tri_ix] = RIGHT_0;
						created_tri_ix--;
						// right side
						tri_edge0[created_tri_ix] = RIGHT_SELF;
						tri_edge1[created_tri_ix] = RIGHT_1;
						tri_edge2[created_tri_ix] = LONGEST_SELF;
						created_tri_ix--;
	/* Scenario #9 */		} else {	// backward scenario
						dprintf("scenario 9\n");
						// left side
						tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
						tri_edge1[created_tri_ix] = LONGEST_1;
						tri_edge2[created_tri_ix] = -RIGHT_1;
						created_tri_ix--;
						// right side
						tri_edge0[created_tri_ix] = RIGHT_ORTHO;
						tri_edge1[created_tri_ix] = -RIGHT_0;
						tri_edge2[created_tri_ix] = LONGEST_SELF;
						created_tri_ix--;
					}
				} else {	// backward scenario
					// first, set the self triangle to left side
					tri_edge0[tid] = -LONGEST_ORTHO;
					tri_edge1[tid] = LEFT_SELF;
					tri_edge2[tid] = -LONGEST_1;
					// split the right side
	/* Scenario #8 */		if (RIGHT > 0) {	// forward scenario
						dprintf("scenario 8\n");
						// left side
						tri_edge0[created_tri_ix] = -RIGHT_SELF;
						tri_edge1[created_tri_ix] = -LONGEST_0;
						tri_edge2[created_tri_ix] = RIGHT_0;
						created_tri_ix--;
						// right side
						tri_edge0[created_tri_ix] = RIGHT_SELF;
						tri_edge1[created_tri_ix] = RIGHT_1;
						tri_edge2[created_tri_ix] = LONGEST_ORTHO;
						created_tri_ix--;
	/* Scenario #10 */		} else {	// backward scenario
						dprintf("scenario 10\n");
						// left side
						tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
						tri_edge1[created_tri_ix] = -LONGEST_0;
						tri_edge2[created_tri_ix] = -RIGHT_1;
						created_tri_ix--;
						// right side
						tri_edge0[created_tri_ix] = RIGHT_ORTHO;
						tri_edge1[created_tri_ix] = -RIGHT_0;
						tri_edge2[created_tri_ix] = LONGEST_ORTHO;
						created_tri_ix--;
					}
				}
			} else {
				if (LONGEST > 0) {	// forward scenario
					// first, set the self triangle to right side
					tri_edge0[tid] = LONGEST_SELF;
					tri_edge1[tid] = LONGEST_1;
					tri_edge2[tid] = RIGHT_SELF;
					// split the left side
	/* Scenario #3 */		if (LEFT > 0) {	// forward scenario
						dprintf("scenario 3\n");
						// left side
						tri_edge0[created_tri_ix] = -LEFT_SELF;
						tri_edge1[created_tri_ix] = -LONGEST_SELF;
						tri_edge2[created_tri_ix] = LEFT_0;
						created_tri_ix--;
						// right side
						tri_edge0[created_tri_ix] = LEFT_SELF;
						tri_edge1[created_tri_ix] = LEFT_1;
						tri_edge2[created_tri_ix] = LONGEST_0;
						created_tri_ix--;
	/* Scenario #5 */		} else {	// backward scenario
						dprintf("scenario 5\n");
						// left side
						tri_edge0[created_tri_ix] = -LEFT_ORTHO;
						tri_edge1[created_tri_ix] = -LONGEST_SELF;
						tri_edge2[created_tri_ix] = -LEFT_1;
						created_tri_ix--;
						// right side
						tri_edge0[created_tri_ix] = LEFT_ORTHO;
						tri_edge1[created_tri_ix] = -LEFT_0;
						tri_edge2[created_tri_ix] = LONGEST_0;
						created_tri_ix--;
					}
				} else {	// backward scenario
					// first, set the self triangle to right side
					tri_edge0[tid] = LONGEST_ORTHO;
					tri_edge1[tid] = -LONGEST_0;
					tri_edge2[tid] = RIGHT_SELF;
					// split the left side
	/* Scenario #4 */		if (LEFT > 0) {	// forward scenario
						dprintf("scenario 4\n");
						// left side
						tri_edge0[created_tri_ix] = -LEFT_SELF;
						tri_edge1[created_tri_ix] = -LONGEST_ORTHO;
						tri_edge2[created_tri_ix] = LEFT_0;
						created_tri_ix--;
						// right side
						tri_edge0[created_tri_ix] = LEFT_SELF;
						tri_edge1[created_tri_ix] = LEFT_1;
						tri_edge2[created_tri_ix] = -LONGEST_1;
						created_tri_ix--;
	/* Scenario #6 */		} else {	// backward scenario
						dprintf("scenario 6\n");
						// left side
						tri_edge0[created_tri_ix] = -LEFT_ORTHO;
						tri_edge1[created_tri_ix] = -LONGEST_ORTHO;
						tri_edge2[created_tri_ix] = -LEFT_1;
						created_tri_ix--;
						// right side
						tri_edge0[created_tri_ix] = LEFT_ORTHO;
						tri_edge1[created_tri_ix] = -LEFT_0;
						tri_edge2[created_tri_ix] = -LONGEST_1;
						created_tri_ix--;
					}
				}
			}
		} else if (marked_count == 3) {
			if (LONGEST > 0) {	// longest forward scenario
				if (LEFT > 0) {	// left forward scenario
	/* Scenario #11 */		if (RIGHT > 0) {	// right forward scenario
						dprintf("scenario 11\n");
						// first, set the self triangle to part 1
						tri_edge0[tid] = -LONGEST_SELF;
						tri_edge1[tid] = LEFT_0;
						tri_edge2[tid] = -LEFT_SELF;
						// part 2
						tri_edge0[created_tri_ix] = LEFT_SELF;
						tri_edge1[created_tri_ix] = LEFT_1;
						tri_edge2[created_tri_ix] = LONGEST_0;
						created_tri_ix--;
						// part 3
						tri_edge0[created_tri_ix] = -RIGHT_SELF;
						tri_edge1[created_tri_ix] = LONGEST_1;
						tri_edge2[created_tri_ix] = RIGHT_0;
						created_tri_ix--;
						// part 4
						tri_edge0[created_tri_ix] = LONGEST_SELF;
						tri_edge1[created_tri_ix] = RIGHT_SELF;
						tri_edge2[created_tri_ix] = RIGHT_1;
						created_tri_ix--;
	/* Scenario #12 */		} else {	// right backward scenario
						dprintf("scenario 12\n");
						// first, set the self triangle to part 1
						tri_edge0[tid] = -LONGEST_SELF;
						tri_edge1[tid] = LEFT_0;
						tri_edge2[tid] = -LEFT_SELF;
						// part 2
						tri_edge0[created_tri_ix] = LEFT_SELF;
						tri_edge1[created_tri_ix] = LEFT_1;
						tri_edge2[created_tri_ix] = LONGEST_0;
						created_tri_ix--;
						// part 3
						tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
						tri_edge1[created_tri_ix] = LONGEST_1;
						tri_edge2[created_tri_ix] = -RIGHT_1;
						created_tri_ix--;
						// part 4
						tri_edge0[created_tri_ix] = LONGEST_SELF;
						tri_edge1[created_tri_ix] = RIGHT_ORTHO;
						tri_edge2[created_tri_ix] = -RIGHT_0;
						created_tri_ix--;
					}
				} else {	// left backward scenario
	/* Scenario #14 */		if (RIGHT > 0) {	// right forward scenario
						dprintf("scenario 14\n");
						// first, set the self triangle to part 1
						tri_edge0[tid] = -LONGEST_SELF;
						tri_edge1[tid] = -LEFT_1;
						tri_edge2[tid] = -LEFT_ORTHO;
						// part 2
						tri_edge0[created_tri_ix] = LEFT_ORTHO;
						tri_edge1[created_tri_ix] = -LEFT_0;
						tri_edge2[created_tri_ix] = LONGEST_0;
						created_tri_ix--;
						// part 3
						tri_edge0[created_tri_ix] = -RIGHT_SELF;
						tri_edge1[created_tri_ix] = LONGEST_1;
						tri_edge2[created_tri_ix] = RIGHT_0;
						created_tri_ix--;
						// part 4
						tri_edge0[created_tri_ix] = LONGEST_SELF;
						tri_edge1[created_tri_ix] = RIGHT_SELF;
						tri_edge2[created_tri_ix] = RIGHT_1;
						created_tri_ix--;
	/* Scenario #13 */		} else {	// right backward scenario
						dprintf("scenario 13\n");
						// first, set the self triangle to part 1
						tri_edge0[tid] = -LONGEST_SELF;
						tri_edge1[tid] = -LEFT_1;
						tri_edge2[tid] = -LEFT_ORTHO;
						// part 2
						tri_edge0[created_tri_ix] = LEFT_ORTHO;
						tri_edge1[created_tri_ix] = -LEFT_0;
						tri_edge2[created_tri_ix] = LONGEST_0;
						created_tri_ix--;
						// part 3
						tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
						tri_edge1[created_tri_ix] = LONGEST_1;
						tri_edge2[created_tri_ix] = -RIGHT_1;
						created_tri_ix--;
						// part 4
						tri_edge0[created_tri_ix] = LONGEST_SELF;
						tri_edge1[created_tri_ix] = RIGHT_ORTHO;
						tri_edge2[created_tri_ix] = -RIGHT_0;
						created_tri_ix--;
					}
				}
			} else {	// longest backward scenario
				if (LEFT > 0) {	// left forward scenario
	/* Scenario #15 */		if (RIGHT > 0) {	// right forward scenario
						dprintf("scenario 15\n");
						// first, set the self triangle to part 1
						tri_edge0[tid] = -LONGEST_ORTHO;
						tri_edge1[tid] = LEFT_0;
						tri_edge2[tid] = -LEFT_SELF;
						// part 2
						tri_edge0[created_tri_ix] = LEFT_SELF;
						tri_edge1[created_tri_ix] = LEFT_1;
						tri_edge2[created_tri_ix] = -LONGEST_1;
						created_tri_ix--;
						// part 3
						tri_edge0[created_tri_ix] = -RIGHT_SELF;
						tri_edge1[created_tri_ix] = -LONGEST_0;
						tri_edge2[created_tri_ix] = RIGHT_0;
						created_tri_ix--;
						// part 4
						tri_edge0[created_tri_ix] = LONGEST_ORTHO;
						tri_edge1[created_tri_ix] = RIGHT_SELF;
						tri_edge2[created_tri_ix] = RIGHT_1;
						created_tri_ix--;
	/* Scenario #16 */		} else {	// right backward scenario
						dprintf("scenario 16\n");
						// first, set the self triangle to part 1
						tri_edge0[tid] = -LONGEST_ORTHO;
						tri_edge1[tid] = LEFT_0;
						tri_edge2[tid] = -LEFT_SELF;
						// part 2
						tri_edge0[created_tri_ix] = LEFT_SELF;
						tri_edge1[created_tri_ix] = LEFT_1;
						tri_edge2[created_tri_ix] = -LONGEST_1;
						created_tri_ix--;
						// part 3
						tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
						tri_edge1[created_tri_ix] = -LONGEST_0;
						tri_edge2[created_tri_ix] = -RIGHT_1;
						created_tri_ix--;
						// part 4
						tri_edge0[created_tri_ix] = LONGEST_ORTHO;
						tri_edge1[created_tri_ix] = RIGHT_ORTHO;
						tri_edge2[created_tri_ix] = -RIGHT_0;
						created_tri_ix--;
					}
				} else {	// left backward scenario
	/* Scenario #18 */		if (RIGHT > 0) {	// right forward scenario
						dprintf("scenario 18\n");
						// first, set the self triangle to part 1
						tri_edge0[tid] = -LONGEST_ORTHO;
						tri_edge1[tid] = -LEFT_1;
						tri_edge2[tid] = -LEFT_ORTHO;
						// part 2
						tri_edge0[created_tri_ix] = LEFT_ORTHO;
						tri_edge1[created_tri_ix] = -LEFT_0;
						tri_edge2[created_tri_ix] = -LONGEST_1;
						created_tri_ix--;
						// part 3
						tri_edge0[created_tri_ix] = -RIGHT_SELF;
						tri_edge1[created_tri_ix] = -LONGEST_0;
						tri_edge2[created_tri_ix] = RIGHT_0;
						created_tri_ix--;
						// part 4
						tri_edge0[created_tri_ix] = LONGEST_ORTHO;
						tri_edge1[created_tri_ix] = RIGHT_SELF;
						tri_edge2[created_tri_ix] = RIGHT_1;
						created_tri_ix--;
	/* Scenario #17 */		} else {	// right backward scenario
						dprintf("scenario 17\n");
						// first, set the self triangle to part 1
						tri_edge0[tid] = -LONGEST_ORTHO;
						tri_edge1[tid] = -LEFT_1;
						tri_edge2[tid] = -LEFT_ORTHO;
						// part 2
						tri_edge0[created_tri_ix] = LEFT_ORTHO;
						tri_edge1[created_tri_ix] = -LEFT_0;
						tri_edge2[created_tri_ix] = -LONGEST_1;
						created_tri_ix--;
						// part 3
						tri_edge0[created_tri_ix] = -RIGHT_ORTHO;
						tri_edge1[created_tri_ix] = -LONGEST_0;
						tri_edge2[created_tri_ix] = -RIGHT_1;
						created_tri_ix--;
						// part 4
						tri_edge0[created_tri_ix] = LONGEST_ORTHO;
						tri_edge1[created_tri_ix] = RIGHT_ORTHO;
						tri_edge2[created_tri_ix] = -RIGHT_0;
						created_tri_ix--;
					}
				}
			}
		}
	}
}

point_t get_opposite_point(int i, point_t *edge_point0, point_t *edge_point1, int *new_edge0, int *new_edge1)
{
	int i_abs = abs(i);
	int ne0, ne1;

	ne0 = new_edge0[i_abs];
	if (ne0 > 0) {	// is it divided?
		if (i > 0) {	// forward
			ne1 = new_edge1[i_abs];
			return edge_point1[abs(ne1)];
		} else {
			return edge_point0[abs(ne0)];
		}
	} else {
		if (i > 0) {
			return edge_point1[i_abs];
		} else {
			return edge_point0[i_abs];
		}
	}
}

void refine_pass2(void)
{
	unsigned int *edge_mark_tree_next = pc_edges.edge_mark_tree_next;
	point_t *edge_point0 = pc_edges.edge_point0;
	point_t *edge_point1 = pc_edges.edge_point1;
	point_t *edge_mid_p = pc_edges.edge_mid_p;
	int *new_edge0 = pc_edges.new_edge0;
	int *new_edge1 = pc_edges.new_edge1;

	PC_FOR_EACH_T {
		unsigned int tid = i;
		unsigned int mn;
		int LONGEST, RIGHT, LEFT;
		int e_ix0, e_ix1, e_ix2, LONGEST_ABS, RIGHT_ABS, LEFT_ABS;

		if (tid >= (pc_tri_count - pc_new_tri_count))
			return;

		e_ix0 = PC_T_E0(tid);
		e_ix1 = PC_T_E1(tid);
		e_ix2 = PC_T_E2(tid);
		LONGEST = PC_T_LE(tid);

		if (LONGEST == abs(e_ix0)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix0;
			RIGHT_ABS = abs(e_ix1);
			RIGHT = e_ix1;
			LEFT_ABS = abs(e_ix2);
			LEFT = e_ix2;
		} else if (LONGEST == abs(e_ix1)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix1;
			RIGHT_ABS = abs(e_ix2);
			RIGHT = e_ix2;
			LEFT_ABS = abs(e_ix0);
			LEFT = e_ix0;
		} else if (LONGEST == abs(e_ix2)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix2;
			RIGHT_ABS = abs(e_ix0);
			RIGHT = e_ix0;
			LEFT_ABS = abs(e_ix1);
			LEFT = e_ix1;
		}

		// process longest
		mn = edge_mark_tree_next[LONGEST_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			if (LONGEST > 0) {	// forward scenario
				edge_point0[LONGEST_ABS] = get_opposite_point(RIGHT, edge_point0, edge_point1, new_edge0, new_edge1);
				edge_point1[LONGEST_ABS] = edge_mid_p[LONGEST_ABS];
			}
		}

		// process right
		mn = edge_mark_tree_next[RIGHT_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			if (RIGHT > 0) {	// forward scenario
				edge_point0[RIGHT_ABS] = edge_mid_p[LONGEST_ABS];
				edge_point1[RIGHT_ABS] = edge_mid_p[RIGHT_ABS];
			}
		}

		// process left
		mn = edge_mark_tree_next[LEFT_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			if (LEFT > 0) {	// forward scenario
				edge_point0[LEFT_ABS] = edge_mid_p[LONGEST_ABS];
				edge_point1[LEFT_ABS] = edge_mid_p[LEFT_ABS];
			}
		}
	}
}

void refine_pass1(void)
{
	unsigned int *edge_mark_tree_next = pc_edges.edge_mark_tree_next;
	point_t *edge_point0 = pc_edges.edge_point0;
	point_t *edge_point1 = pc_edges.edge_point1;
	point_t *edge_mid_p = pc_edges.edge_mid_p;
	int *new_edge0 = pc_edges.new_edge0;
	int *new_edge1 = pc_edges.new_edge1;
	int *orth_edge = pc_edges.orth_edge;

	PC_FOR_EACH_T {
		unsigned int tid = i;
		unsigned int created_edge_ix = pc_edge_count - pc_new_edge_count;
		unsigned int mn;
		int LONGEST, RIGHT, LEFT;
		int e_ix0, e_ix1, e_ix2, LONGEST_ABS, RIGHT_ABS, LEFT_ABS;
		unsigned int cnt, rcnt;

		if (tid >= (pc_tri_count - pc_new_tri_count))
			return;

		cnt = PC_T_CNT_SCAN(tid);
		rcnt = PC_T_RCNT_SCAN(tid);
		created_edge_ix += 2*__GET_T_CNT(cnt) + __GET_T_RCNT(rcnt) - 1;
		e_ix0 = PC_T_E0(tid);
		e_ix1 = PC_T_E1(tid);
		e_ix2 = PC_T_E2(tid);
		LONGEST = PC_T_LE(tid);

		if (LONGEST == abs(e_ix0)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix0;
			RIGHT_ABS = abs(e_ix1);
			RIGHT = e_ix1;
			LEFT_ABS = abs(e_ix2);
			LEFT = e_ix2;
		} else if (LONGEST == abs(e_ix1)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix1;
			RIGHT_ABS = abs(e_ix2);
			RIGHT = e_ix2;
			LEFT_ABS = abs(e_ix0);
			LEFT = e_ix0;
		} else if (LONGEST == abs(e_ix2)) {
			LONGEST_ABS = LONGEST;
			LONGEST = e_ix2;
			RIGHT_ABS = abs(e_ix0);
			RIGHT = e_ix0;
			LEFT_ABS = abs(e_ix1);
			LEFT = e_ix1;
		}

		// process longest
		mn = edge_mark_tree_next[LONGEST_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			if (LONGEST > 0) {	// forward scenario
				edge_point0[created_edge_ix] = edge_point0[LONGEST_ABS];
				edge_point1[created_edge_ix] = edge_mid_p[LONGEST_ABS];
				new_edge0[LONGEST_ABS] = created_edge_ix;
				created_edge_ix--;
				edge_point0[created_edge_ix] = edge_mid_p[LONGEST_ABS];
				edge_point1[created_edge_ix] = edge_point1[LONGEST_ABS];
				new_edge1[LONGEST_ABS] = created_edge_ix;
				created_edge_ix--;
			} else {
				edge_point0[created_edge_ix] = (RIGHT > 0) ? edge_point1[RIGHT_ABS] : edge_point0[RIGHT_ABS];
				edge_point1[created_edge_ix] = edge_mid_p[LONGEST_ABS];
				orth_edge[LONGEST_ABS] = created_edge_ix;
				created_edge_ix--;
			}
		}

		// process right
		mn = edge_mark_tree_next[RIGHT_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			if (RIGHT > 0) {	// forward scenario
				edge_point0[created_edge_ix] = edge_point0[RIGHT_ABS];
				edge_point1[created_edge_ix] = edge_mid_p[RIGHT_ABS];
				new_edge0[RIGHT_ABS] = created_edge_ix;
				created_edge_ix--;
				edge_point0[created_edge_ix] = edge_mid_p[RIGHT_ABS];
				edge_point1[created_edge_ix] = edge_point1[RIGHT_ABS];
				new_edge1[RIGHT_ABS] = created_edge_ix;
				created_edge_ix--;
			} else {
				edge_point0[created_edge_ix] = edge_mid_p[LONGEST_ABS];
				edge_point1[created_edge_ix] = edge_mid_p[RIGHT_ABS];
				orth_edge[RIGHT_ABS] = created_edge_ix;
				created_edge_ix--;
			}
		}

		// process left
		mn = edge_mark_tree_next[LEFT_ABS];
		if ((__GET_E_MARK(mn) & 2)) {
			if (LEFT > 0) {	// forward scenario
				edge_point0[created_edge_ix] = edge_point0[LEFT_ABS];
				edge_point1[created_edge_ix] = edge_mid_p[LEFT_ABS];
				new_edge0[LEFT_ABS] = created_edge_ix;
				created_edge_ix--;
				edge_point0[created_edge_ix] = edge_mid_p[LEFT_ABS];
				edge_point1[created_edge_ix] = edge_point1[LEFT_ABS];
				new_edge1[LEFT_ABS] = created_edge_ix;
				created_edge_ix--;
			} else {
				edge_point0[created_edge_ix] = edge_mid_p[LONGEST_ABS];
				edge_point1[created_edge_ix] = edge_mid_p[LEFT_ABS];
				orth_edge[LEFT_ABS] = created_edge_ix;
				created_edge_ix--;
			}
		}
	}
}

void pc_refine(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;

	printf(DGREEN"[%s]"NORM" WORK STARTED\n", __func__);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

	refine_pass1();

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);
	printf(DGREEN"[%s]"NORM" PASS1 TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	refine_pass2();

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);
	printf(DGREEN"[%s]"NORM" PASS2 TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	refine_pass3();

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);
	printf(DGREEN"[%s]"NORM" PASS3 TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);
}

int pc_realloc_tris(void)
{
	pc_tris.tri_edge0 = (int *)realloc(pc_tris.tri_edge0, (pc_tri_count + pc_new_tri_count)*sizeof(int));
	if (pc_tris.tri_edge0 == NULL) {
		printf("cannot realloc pc_tris.tri_edge0!\n");
		return -1;
	}

	pc_tris.tri_edge1 = (int *)realloc(pc_tris.tri_edge1, (pc_tri_count + pc_new_tri_count)*sizeof(int));
	if (pc_tris.tri_edge1 == NULL) {
		printf("cannot realloc pc_tris.tri_edge1!\n");
		return -1;
	}

	pc_tris.tri_edge2 = (int *)realloc(pc_tris.tri_edge2, (pc_tri_count + pc_new_tri_count)*sizeof(int));
	if (pc_tris.tri_edge2 == NULL) {
		printf("cannot realloc pc_tris.tri_edge2!\n");
		return -1;
	}

	pc_tris.tri_longest_edge = (int *)realloc(pc_tris.tri_longest_edge, (pc_tri_count + pc_new_tri_count)*sizeof(int));
	if (pc_tris.tri_longest_edge == NULL) {
		printf("cannot realloc pc_tris.tri_longest_edge!\n");
		return -1;
	}

	pc_tris.tri_counter = (unsigned int *)realloc(pc_tris.tri_counter, (pc_tri_count + pc_new_tri_count)*sizeof(int));
	if (pc_tris.tri_counter == NULL) {
		printf("cannot realloc pc_tris.tri_counter!\n");
		return -1;
	}

	pc_tris.tri_counter_scan = (unsigned int *)realloc(pc_tris.tri_counter_scan, (pc_tri_count + pc_new_tri_count)*sizeof(int));
	if (pc_tris.tri_counter_scan == NULL) {
		printf("cannot realloc pc_tris.tri_counter_scan!\n");
		return -1;
	}

	pc_tris.tri_rev_counter = (unsigned int *)realloc(pc_tris.tri_rev_counter, (pc_tri_count + pc_new_tri_count)*sizeof(int));
	if (pc_tris.tri_rev_counter == NULL) {
		printf("cannot realloc pc_tris.tri_rev_counter!\n");
		return -1;
	}

	pc_tris.tri_rev_counter_scan = (unsigned int *)realloc(pc_tris.tri_rev_counter_scan, (pc_tri_count + pc_new_tri_count)*sizeof(int));
	if (pc_tris.tri_rev_counter_scan == NULL) {
		printf("cannot realloc pc_tris.tri_rev_counter_scan!\n");
		return -1;
	}

	return 0;
}

int pc_realloc_edges(void)
{
	pc_edges.edge_point0 = (point_t *)realloc(pc_edges.edge_point0, (pc_edge_count + pc_new_edge_count)*sizeof(point_t));
	if (pc_edges.edge_point0 == NULL) {
		printf("cannot realloc edge_point0 arr!\n");
		return -1;
	}

	pc_edges.edge_point1 = (point_t *)realloc(pc_edges.edge_point1, (pc_edge_count + pc_new_edge_count)*sizeof(point_t));
	if (pc_edges.edge_point1 == NULL) {
		printf("cannot realloc edge_point1 arr!\n");
		return -1;
	}

	pc_edges.edge_mid_p = (point_t *)realloc(pc_edges.edge_mid_p, (pc_edge_count + pc_new_edge_count)*sizeof(point_t));
	if (pc_edges.edge_mid_p == NULL) {
		printf("cannot realloc edge_mid_p arr!\n");
		return -1;
	}

	pc_edges.edge_len = (float *)realloc(pc_edges.edge_len, (pc_edge_count + pc_new_edge_count)*sizeof(float));
	if (pc_edges.edge_len == NULL) {
		printf("cannot realloc edge_len arr!\n");
		return -1;
	}

	pc_edges.edge_mark_tree_next = (unsigned int *)realloc(pc_edges.edge_mark_tree_next, (pc_edge_count + pc_new_edge_count)*sizeof(int));
	if (pc_edges.edge_mark_tree_next == NULL) {
		printf("cannot realloc edge_mark_tree_next arr!\n");
		return -1;
	}

	pc_edges.new_edge0 = (int *)realloc(pc_edges.new_edge0, (pc_edge_count + pc_new_edge_count)*sizeof(int));
	if (pc_edges.new_edge0 == NULL) {
		printf("cannot realloc new_edge0 arr!\n");
		return -1;
	}

	pc_edges.new_edge1 = (int *)realloc(pc_edges.new_edge1, (pc_edge_count + pc_new_edge_count)*sizeof(int));
	if (pc_edges.new_edge1 == NULL) {
		printf("cannot realloc new_edge1 arr!\n");
		return -1;
	}

	pc_edges.orth_edge = (int *)realloc(pc_edges.orth_edge, (pc_edge_count + pc_new_edge_count)*sizeof(int));
	if (pc_edges.orth_edge == NULL) {
		printf("cannot realloc orth_edge arr!\n");
		return -1;
	}

	return 0;
}

void pc_create_new_elem_arrs(void)
{
	pc_new_edge_count = 2*__GET_T_CNT(pc_newelemcount) + __GET_T_RCNT(pc_newelemcount_rev);
	pc_new_tri_count = __GET_T_CNT(pc_newelemcount) + __GET_T_RCNT(pc_newelemcount_rev);

	printf("new_edge_count=%d new_tri_count=%d\n", pc_new_edge_count, pc_new_tri_count);

	if (pc_realloc_edges() < 0) {
		printf("host realloc error\n");
		exit(1);
	}

	if (pc_realloc_tris() < 0) {
		printf("host realloc error\n");
		exit(1);
	}

	pc_edge_count += pc_new_edge_count;
	pc_tri_count += pc_new_tri_count;
}

void pc_prefix_counters(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;

	printf(DGREEN"[%s]"NORM" WORK STARTED\n", __func__);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

	PC_FOR_EACH_T {
		if (!i)
			continue;
	
		PC_T_CNT(i) += PC_T_CNT(i-1);
		PC_T_RCNT(i) += PC_T_RCNT(i-1);
	}

	pc_newelemcount = PC_T_CNT(pc_tri_count-1);
	pc_newelemcount_rev = PC_T_RCNT(pc_tri_count-1);

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// bilal: workaround: since we cannot get the scan directly to tri_counter_scan
	memcpy(pc_tris.tri_counter_scan, pc_tris.tri_counter, pc_tri_count*sizeof(int));
	memcpy(pc_tris.tri_rev_counter_scan, pc_tris.tri_rev_counter, pc_tri_count*sizeof(int));
}

void pc_get_counters(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;

	printf(DGREEN"[%s]"NORM" WORK STARTED\n", __func__);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

	PC_FOR_EACH_T {
		int p_counter = 0, p_rev_counter = 0, e_ix;
		unsigned int mn;

		e_ix = PC_T_E0(i);
		mn = PC_E_MN(abs(e_ix));
		if ((__GET_E_MARK(mn) & 2)) {
			if (e_ix >= 0)
				p_counter++;
			else
				p_rev_counter++;
		}

		e_ix = PC_T_E1(i);
		mn = PC_E_MN(abs(e_ix));
		if ((__GET_E_MARK(mn) & 2)) {
			if (e_ix >= 0)
				p_counter++;
			else
				p_rev_counter++;
		}

		e_ix = PC_T_E2(i);
		mn = PC_E_MN(abs(e_ix));
		if ((__GET_E_MARK(mn) & 2)) {
			if (e_ix >= 0)
				p_counter++;
			else
				p_rev_counter++;
		}

		PC_T_CNT(i) = p_counter;
		PC_T_RCNT(i) = p_rev_counter;
	}

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);
}

void pc_follow_links(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	unsigned int *edge_mark_tree_next = pc_edges.edge_mark_tree_next;

	printf(DGREEN"[%s]"NORM" WORK STARTED\n", __func__);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

	PC_FOR_EACH_E {
		unsigned int mn, next_mn;
		int next_ix;
		int tid = i;

		mn = edge_mark_tree_next[tid];
		if (!(__GET_E_MARK(mn) & 2)) {	// not marked
			continue;
		}

		next_ix = __GET_E_NEXT(mn);
		if (!next_ix || (next_ix == NO_LINK)) {
			continue;
		}

		while (next_ix && (next_ix != NO_LINK)) {
			next_mn = edge_mark_tree_next[next_ix];
			if (__GET_E_MARK(next_mn) & 2) {	// next is already marked
				break;
			}
			__SET_E_NEXT(mn, __GET_E_NEXT(next_mn));
			next_mn |= 2;
			__SET_E_NEXT(next_mn, NO_LINK);
			edge_mark_tree_next[next_ix] = next_mn;
			next_ix = __GET_E_NEXT(mn);
		}

		__SET_E_NEXT(mn, NO_LINK);
		edge_mark_tree_next[tid] = mn;
	}

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);
}

void pc_correct_marks(void)
{
	PC_FOR_EACH_E {
		if (PC_E_LEN(i) < 0)
			PC_E_MN(i) |= 2;
	}
}

void pc_establish_links(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;

	printf(DGREEN"[%s]"NORM" WORK STARTED\n", __func__);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

	PC_FOR_EACH_T {
		int edge, ledge;
		unsigned int mn;

		ledge = PC_T_LE(i);

		edge = abs(PC_T_E0(i));
		if (edge != ledge) {
			mn = PC_E_MN(edge);
			if (__GET_E_MARK(mn) & 1) {
				__SET_E_NEXT(mn, ledge);
				PC_E_MN(edge) = mn;
			}
		}

		edge = abs(PC_T_E1(i));
		if (edge != ledge) {
			mn = PC_E_MN(edge);
			if (__GET_E_MARK(mn) & 1) {
				__SET_E_NEXT(mn, ledge);
				PC_E_MN(edge) = mn;
			}
		}

		edge = abs(PC_T_E2(i));
		if (edge != ledge) {
			mn = PC_E_MN(edge);
			if (__GET_E_MARK(mn) & 1) {
				__SET_E_NEXT(mn, ledge);
				PC_E_MN(edge) = mn;
			}
		}
	}

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);
}

void pc_mark_longest_edges(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;

	printf(DGREEN"[%s]"NORM" WORK STARTED\n", __func__);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

	PC_FOR_EACH_T {
		float llen = 0, clen;
		int ledge;
		int edge_ix;

		edge_ix = abs(PC_T_E0(i));
		clen = abs(PC_E_LEN(edge_ix));
		if (clen > llen) {
			llen = clen;
			ledge = edge_ix;
		}

		edge_ix = abs(PC_T_E1(i));
		clen = abs(PC_E_LEN(edge_ix));
		if (clen > llen) {
			llen = clen;
			ledge = edge_ix;
		}

		edge_ix = abs(PC_T_E2(i));
		clen = abs(PC_E_LEN(edge_ix));
		if (clen > llen) {
			llen = clen;
			ledge = edge_ix;
		}

		if (PC_T_LE(i) == 1) {
			PC_E_LEN(ledge) = -llen;	// negative edge len is blackmark
		}

		PC_E_MN(ledge) = 1;		// mark as longest edge
		PC_T_LE(i) = ledge;
	}

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);
}

static inline point_t calc_mid_point(point_t *p1, point_t *p2)
{
	point_t ret;

	ret.x = (p1->x + p2->x) / 2;
	ret.y = (p1->y + p2->y) / 2;

	return ret;
}

void pc_calc_edge_lengths_mid_p(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;

	printf(DGREEN"[%s]"NORM" WORK STARTED\n", __func__);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

	PC_FOR_EACH_E {
		point_t p0, p1, mid_p;
		float len;

		p0 = PC_E_P0(i);
		p1 = PC_E_P1(i);

		len = sqrtf(powf(fabs(p0.x - p1.x), 2) + powf(fabs(p0.y - p1.y), 2));
		mid_p = calc_mid_point(&p0, &p1);

		PC_E_LEN(i) = len;
		PC_E_MIDP(i) = mid_p;
	}

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);
}

void pc_print_input(void)
{
	printf("Edges:\n");
	PC_FOR_EACH_E {
		printf("edge[%d]=[%f,%f]\t[%f,%f]\tmid=[%f,%f]\tlen=%f\tmark=%d\tnext=%d\tnew_edges={%d, %d, orth=%d}\n", i, PC_E_P0(i).x, PC_E_P0(i).y, PC_E_P1(i).x, PC_E_P1(i).y, PC_E_MIDP(i).x, PC_E_MIDP(i).y,  PC_E_LEN(i), PC_GET_E_MARK(i), (PC_GET_E_NEXT(i) != NO_LINK) ? (int)PC_GET_E_NEXT(i) : -1, PC_E_NE0(i), PC_E_NE1(i), PC_E_OE(i));
	}

	printf("Triangles:\n");
	PC_FOR_EACH_T {
		printf("triangle[%d]=%d %d %d longest=%d\tcounter=%d rev_counter=%d\tcounter_scan=%d counter_scan_rev=%d\n", i, PC_T_E0(i), PC_T_E1(i), PC_T_E2(i), PC_T_LE(i), PC_GET_T_CNT(i), PC_GET_T_RCNT(i), PC_GET_T_CNT_SCAN(i), PC_GET_T_RCNT_SCAN(i));
	}

	return;
}

int mesh_refine_pc(int only_pc)
{
	printf("PC Starting process...\n");

#if 1
	if (debug)
		pc_print_input();
#endif

	// bilal: workaround for floating point problem! skip step 1 if the code runs on device
	if (only_pc) {
		// step 1: calculate edge lengths and mid points
		pc_calc_edge_lengths_mid_p();
	}

	// step 2: mark longest edge of each triangle
	pc_mark_longest_edges();

	// step 3: establish links
	pc_establish_links();

	// step 4: follow links
	pc_correct_marks();
	pc_follow_links();

	// step 5: get the initial value of counters
	pc_get_counters();

	// step 6: prefix the counters
	pc_prefix_counters();

	pc_create_new_elem_arrs();

	// step 7: refine the mesh: create new edges
	pc_refine();

#if 1
	if (debug)
		pc_print_input();
#endif

	return 0;
}

int compare_results(void)
{
	int ret = 0;

	// sanity checks

	// check triangles having valid edge
	PC_FOR_EACH_T {
		if ((PC_T_E0(i) == 0) || (abs(PC_T_E0(i)) >= pc_edge_count) || (PC_T_E1(i) == 0) || (abs(PC_T_E1(i)) >= pc_edge_count) || (PC_T_E2(i) == 0) || (abs(PC_T_E2(i)) >= pc_edge_count)) {
			printf("PC sanity failed on triangle: %d\n", i);
			ret = -1;
		}
	}

	FOR_EACH_T {
		if ((T_E0(i) == 0) || (abs(T_E0(i)) >= edge_count) || (T_E1(i) == 0) || (abs(T_E1(i)) >= edge_count) || (T_E2(i) == 0) || (abs(T_E2(i)) >= edge_count)) {
			printf("sanity failed on triangle: %d\n", i);
			ret = -1;
		}
	}

	// check mark and next consistency
	FOR_EACH_E {
		if (i >= (edge_count - new_edge_count))
			break;
		if ((GET_E_MARK(i) == 3) && (GET_E_NEXT(i) != NO_LINK) && (GET_E_NEXT(i) != 0)) {
			printf("sanity failed on edge: %d mark=%d next=%d\n", i, GET_E_MARK(i), GET_E_NEXT(i));
		}
	}

	FOR_EACH_E {
		if (i >= (pc_edge_count - pc_new_edge_count))
			break;
		if ((PC_GET_E_MARK(i) == 3) && (PC_GET_E_NEXT(i) != NO_LINK) && (PC_GET_E_NEXT(i) != 0)) {
			printf("PC sanity failed on edge: %d mark=%d next=%d\n", i, PC_GET_E_MARK(i), PC_GET_E_NEXT(i));
		}
	}

	// check counter have a valid value
	FOR_EACH_T {
		if (!i)
			continue;
		if (i >= (pc_tri_count - pc_new_tri_count))
			break;
		if (((PC_T_CNT(i) + PC_T_RCNT(i)) - (PC_T_CNT(i-1) + PC_T_RCNT(i-1))) > 3) {
			printf("PC sanity failed on triangle: %d\n", i);
		}
	}

	FOR_EACH_T {
		if (i >= (tri_count - new_tri_count))
			break;
		if ((T_CNT(i) + T_RCNT(i)) > 3) {
			printf("sanity failed on triangle: %d\n", i);
		}
	}

	// comparisons

	if (pc_edge_count != edge_count) {
		printf("edge count mismatch: %d != %d\n", pc_edge_count, edge_count);
		return -1;
	}

	if (pc_tri_count != tri_count) {
		printf("tri count mismatch: %d != %d\n", pc_tri_count, tri_count);
		return -1;
	}

	if (pc_new_edge_count != new_edge_count) {
		printf("new edge count mismatch: %d != %d\n", pc_new_edge_count, new_edge_count);
		return -1;
	}

	if (pc_new_tri_count != new_tri_count) {
		printf("new tri count mismatch: %d != %d\n", pc_new_tri_count, new_tri_count);
		return -1;
	}

	FOR_EACH_E {
		if ((PC_E_P0(i).x != E_P0(i).x) || (PC_E_P0(i).y != E_P0(i).y) || (PC_E_P1(i).x != E_P1(i).x) || (PC_E_P1(i).y != E_P1(i).y)) {
			printf("edge mismatch on %d\n", i);
			ret = -1;
		}
	}

	FOR_EACH_T {
		if ((PC_T_E0(i) != T_E0(i)) || (PC_T_E1(i) != T_E1(i)) || (PC_T_E2(i) != T_E2(i)) /*|| (T(i).er.longest_edge_ix != CUDA_T(i).er.longest_edge_ix)*/) {
			printf("triangle mismatch on %d\n", i);
			ret = -1;
		}
	}

	return ret;
}

void pc_copy_tris(void)
{
	memcpy(pc_tris.tri_edge0, tris.tri_edge0, sizeof(int)*tri_count);
	memcpy(pc_tris.tri_edge1, tris.tri_edge1, sizeof(int)*tri_count);
	memcpy(pc_tris.tri_edge2, tris.tri_edge2, sizeof(int)*tri_count);
	memcpy(pc_tris.tri_longest_edge, tris.tri_longest_edge, sizeof(int)*tri_count);
	memcpy(pc_tris.tri_counter, tris.tri_counter, sizeof(int)*tri_count);
	memcpy(pc_tris.tri_counter_scan, tris.tri_counter_scan, sizeof(int)*tri_count);
	memcpy(pc_tris.tri_rev_counter, tris.tri_rev_counter, sizeof(int)*tri_count);
	memcpy(pc_tris.tri_rev_counter_scan, tris.tri_rev_counter_scan, sizeof(int)*tri_count);
}

int pc_alloc_tris(void)
{
	pc_tri_count = tri_count;

	pc_tris.tri_edge0 = (int *)calloc(pc_tri_count, sizeof(int));
	if (pc_tris.tri_edge0 == NULL) {
		printf("cannot alloc tris.tri_edge0!\n");
		return -1;
	}

	pc_tris.tri_edge1 = (int *)calloc(pc_tri_count, sizeof(int));
	if (pc_tris.tri_edge1 == NULL) {
		printf("cannot alloc tris.tri_edge1!\n");
		return -1;
	}

	pc_tris.tri_edge2 = (int *)calloc(pc_tri_count, sizeof(int));
	if (pc_tris.tri_edge2 == NULL) {
		printf("cannot alloc tris.tri_edge2!\n");
		return -1;
	}

	pc_tris.tri_longest_edge = (int *)calloc(pc_tri_count, sizeof(int));
	if (pc_tris.tri_longest_edge == NULL) {
		printf("cannot alloc tris.tri_longest_edge!\n");
		return -1;
	}

	pc_tris.tri_counter = (unsigned int *)calloc(pc_tri_count, sizeof(int));
	if (pc_tris.tri_counter == NULL) {
		printf("cannot alloc tris.tri_counter!\n");
		return -1;
	}

	pc_tris.tri_counter_scan = (unsigned int *)calloc(pc_tri_count, sizeof(int));
	if (pc_tris.tri_counter_scan == NULL) {
		printf("cannot alloc tris.tri_counter_scan!\n");
		return -1;
	}

	pc_tris.tri_rev_counter = (unsigned int *)calloc(pc_tri_count, sizeof(int));
	if (pc_tris.tri_rev_counter == NULL) {
		printf("cannot alloc tris.tri_rev_counter!\n");
		return -1;
	}

	pc_tris.tri_rev_counter_scan = (unsigned int *)calloc(pc_tri_count, sizeof(int));
	if (pc_tris.tri_rev_counter_scan == NULL) {
		printf("cannot alloc tris.tri_rev_counter_scan!\n");
		return -1;
	}

	return 0;
}

void pc_copy_edges(void)
{
	memcpy(pc_edges.edge_point0, edges.edge_point0, sizeof(point_t)*edge_count);
	memcpy(pc_edges.edge_point1, edges.edge_point1, sizeof(point_t)*edge_count);
	memcpy(pc_edges.edge_mid_p, edges.edge_mid_p, sizeof(point_t)*edge_count);
	memcpy(pc_edges.edge_len, edges.edge_len, sizeof(float)*edge_count);
	memcpy(pc_edges.edge_mark_tree_next, edges.edge_mark_tree_next, sizeof(int)*edge_count);
	memcpy(pc_edges.new_edge0, edges.new_edge0, sizeof(int)*edge_count);
	memcpy(pc_edges.new_edge1, edges.new_edge1, sizeof(int)*edge_count);
	memcpy(pc_edges.orth_edge, edges.orth_edge, sizeof(int)*edge_count);
}

int pc_alloc_edges(void)
{
	pc_edge_count = edge_count;

	pc_edges.edge_point0 = (point_t *)calloc(pc_edge_count, sizeof(point_t));
	if (pc_edges.edge_point0 == NULL) {
		printf("cannot alloc edge_point0 arr!\n");
		return -1;
	}

	pc_edges.edge_point1 = (point_t *)calloc(pc_edge_count, sizeof(point_t));
	if (pc_edges.edge_point1 == NULL) {
		printf("cannot alloc edge_point1 arr!\n");
		return -1;
	}

	pc_edges.edge_mid_p = (point_t *)calloc(pc_edge_count, sizeof(point_t));
	if (pc_edges.edge_mid_p == NULL) {
		printf("cannot alloc edge_mid_p arr!\n");
		return -1;
	}

	pc_edges.edge_len = (float *)calloc(pc_edge_count, sizeof(float));
	if (pc_edges.edge_len == NULL) {
		printf("cannot alloc edge_len arr!\n");
		return -1;
	}

	pc_edges.edge_mark_tree_next = (unsigned int *)calloc(pc_edge_count, sizeof(int));
	if (pc_edges.edge_mark_tree_next == NULL) {
		printf("cannot alloc edge_mark_tree_next arr!\n");
		return -1;
	}

	pc_edges.new_edge0 = (int *)calloc(pc_edge_count, sizeof(int));
	if (pc_edges.new_edge0 == NULL) {
		printf("cannot alloc new_edge0 arr!\n");
		return -1;
	}

	pc_edges.new_edge1 = (int *)calloc(pc_edge_count, sizeof(int));
	if (pc_edges.new_edge1 == NULL) {
		printf("cannot alloc new_edge1 arr!\n");
		return -1;
	}

	pc_edges.orth_edge = (int *)calloc(pc_edge_count, sizeof(int));
	if (pc_edges.orth_edge == NULL) {
		printf("cannot alloc orth_edge arr!\n");
		return -1;
	}

	return 0;
}

int pc_alloc_copy_input(void)
{
	if (pc_alloc_edges() < 0) {
		printf("pc alloc error\n");
		return -1;
	}

	pc_copy_edges();

	if (pc_alloc_tris() < 0) {
		printf("pc alloc error\n");
		return -1;
	}

	pc_copy_tris();

	return 0;
}

int pc_write_edges(FILE *outfile)
{
	printf("Writing %d edges... ", pc_edge_count - 1);
	fflush(stdout);

	fprintf(outfile, "#edges\n");
	fprintf(outfile, "%d\n", pc_edge_count - 1);

	PC_FOR_EACH_E {
		fprintf(outfile, "[%.17g %.17g] \t[%.17g %.17g]\n", PC_E_P0(i).x, PC_E_P0(i).y, PC_E_P1(i).x, PC_E_P1(i).y);
	}

	fprintf(outfile, "\n");

	printf("OK, ");
	return 0;
}

int pc_write_triangles(FILE *outfile)
{
	printf("Writing %d triangles... ", pc_tri_count);
	fflush(stdout);

	fprintf(outfile, "#triangles\n");
	fprintf(outfile, "%d\n", pc_tri_count);

	PC_FOR_EACH_T {
		// TODO: refine edilenleri bul
		fprintf(outfile, "%d \t%d \t%d \t%d\n", PC_T_E0(i), PC_T_E1(i), PC_T_E2(i), 0);
	}

	fprintf(outfile, "\n");

	printf("OK, ");
	return 0;
}

int pc_write_output(FILE *outfile)
{
	printf("Generating PC output file... ");
	fflush(stdout);

#if 0
	if (write_points(outfile) < 0) {
		printf("error on writing points!\n");
		return -1;
	}
#endif
	if (pc_write_edges(outfile) < 0) {
		printf("error on writing edges!\n");
		return -1;
	}

	if (pc_write_triangles(outfile) < 0) {
		printf("error on writing triangles!\n");
		return -1;
	}

	printf("DONE!\n");

	return 0;
}

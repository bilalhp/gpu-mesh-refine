#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <sys/time.h>

#include <main.h>

#define MAX_LINE_SIZE		1024

#define TOLERANCE			0.000001
#define COMPARE_FLOAT2(_f1, _f2)	(fabs(_f1 - _f2) > TOLERANCE)

typedef struct tri_point	tri_point_t;
typedef struct in_edge		in_edge_t;

struct tri_point {
	int point[3];
};

struct in_edge {
	int point_ix[2];
	int next;
};

tri_point_t *tri_point_arr;
in_edge_t *in_edge_arr;

unsigned int in_edge_count;


int get_rand_int(int min, int max)
{
	return min+(rand()%(max - min));
}

int copy_back_tris(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;

	gettimeofday(&start_time, NULL);

	cudaSafeCall(hipMemcpy(tris.tri_edge0, d_tris->tri_edge0, sizeof(int)*tri_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(tris.tri_edge1, d_tris->tri_edge1, sizeof(int)*tri_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(tris.tri_edge2, d_tris->tri_edge2, sizeof(int)*tri_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(tris.tri_longest_edge, d_tris->tri_longest_edge, sizeof(int)*tri_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(tris.tri_counter, d_tris->tri_counter, sizeof(int)*tri_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(tris.tri_counter_scan, d_tris->tri_counter_scan, sizeof(int)*tri_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(tris.tri_rev_counter, d_tris->tri_rev_counter, sizeof(int)*tri_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(tris.tri_rev_counter_scan, d_tris->tri_rev_counter_scan, sizeof(int)*tri_count, hipMemcpyDeviceToHost));

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf("[%s] MEMCPY TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	return 0;
}

int copy_back_edges(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;

	gettimeofday(&start_time, NULL);

	cudaSafeCall(hipMemcpy(edges.edge_point0, d_edges->edge_point0, sizeof(point_t)*edge_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(edges.edge_point1, d_edges->edge_point1, sizeof(point_t)*edge_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(edges.edge_mid_p, d_edges->edge_mid_p, sizeof(point_t)*edge_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(edges.edge_len, d_edges->edge_len, sizeof(float)*edge_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(edges.edge_mark_tree_next, d_edges->edge_mark_tree_next, sizeof(int)*edge_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(edges.new_edge0, d_edges->new_edge0, sizeof(int)*edge_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(edges.new_edge1, d_edges->new_edge1, sizeof(int)*edge_count, hipMemcpyDeviceToHost));
	cudaSafeCall(hipMemcpy(edges.orth_edge, d_edges->orth_edge, sizeof(int)*edge_count, hipMemcpyDeviceToHost));

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf("[%s] MEMCPY TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	return 0;
}

int copy_back(void)
{
	copy_back_edges();
	copy_back_tris();

	return 0;
}

void device_cleanup(void)
{
	if (d_edges->edge_point0) {
		cudaSafeCall(hipFree((void*)d_edges->edge_point0));
		cudaSafeCall(hipFree((void*)d_edges->edge_point1));
		cudaSafeCall(hipFree((void*)d_edges->edge_mid_p));
		cudaSafeCall(hipFree((void*)d_edges->edge_len));
		cudaSafeCall(hipFree((void*)d_edges->edge_mark_tree_next));
		cudaSafeCall(hipFree((void*)d_edges->new_edge0));
		cudaSafeCall(hipFree((void*)d_edges->new_edge1));
		cudaSafeCall(hipFree((void*)d_edges->orth_edge));
		d_edges->edge_point0 = NULL;
	}

	if (d_tris->tri_edge0) {
		cudaSafeCall(hipFree((void*)d_tris->tri_edge0));
		cudaSafeCall(hipFree((void*)d_tris->tri_edge1));
		cudaSafeCall(hipFree((void*)d_tris->tri_edge2));
		cudaSafeCall(hipFree((void*)d_tris->tri_longest_edge));
		cudaSafeCall(hipFree((void*)d_tris->tri_counter));
		cudaSafeCall(hipFree((void*)d_tris->tri_counter_scan));
		cudaSafeCall(hipFree((void*)d_tris->tri_rev_counter));
		cudaSafeCall(hipFree((void*)d_tris->tri_rev_counter_scan));
		d_tris->tri_edge0 = NULL;
	}
}

int malloc_copy_input_to_device(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	unsigned int edge_size, tri_size, copy_edge_size, copy_tri_size;

	edge_size = 3*sizeof(point_t)*edge_count + sizeof(float)*edge_count + 4*sizeof(int)*edge_count;
	tri_size = 8*sizeof(int)*tri_count;
	copy_edge_size = 2*sizeof(point_t)*edge_count + 2*sizeof(int)*edge_count;
	copy_tri_size = 4*sizeof(int)*tri_count;

	printf("Total allocated memory = %u + %u bytes [%.1lf MB]\n", edge_size, tri_size, (double)(edge_size + tri_size)/(1024*1024));

	cudaSafeCall(hipMalloc((void**)&d_edges->edge_point0, sizeof(point_t)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->edge_point1, sizeof(point_t)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->edge_mid_p, sizeof(point_t)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->edge_len, sizeof(float)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->edge_mark_tree_next, sizeof(int)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->new_edge0, sizeof(int)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->new_edge1, sizeof(int)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->orth_edge, sizeof(int)*edge_count));

	cudaSafeCall(hipMalloc((void**)&d_tris->tri_edge0, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_edge1, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_edge2, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_longest_edge, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_counter, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_counter_scan, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_rev_counter, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_rev_counter_scan, sizeof(int)*tri_count));

	gettimeofday(&start_time, NULL);

	cudaSafeCall(hipMemcpy(d_tris->tri_edge0, tris.tri_edge0, sizeof(int)*tri_count, hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_edge1, tris.tri_edge1, sizeof(int)*tri_count, hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_edge2, tris.tri_edge2, sizeof(int)*tri_count, hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_longest_edge, tris.tri_longest_edge, sizeof(int)*tri_count, hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_tris->tri_counter, tris.tri_counter, sizeof(int)*tri_count, hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_tris->tri_counter_scan, tris.tri_counter_scan, sizeof(int)*tri_count, hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_tris->tri_rev_counter, tris.tri_rev_counter, sizeof(int)*tri_count, hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_tris->tri_rev_counter_scan, tris.tri_rev_counter_scan, sizeof(int)*tri_count, hipMemcpyHostToDevice));

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf("[%s] TRIS MEMCPY TIME ELAPSED = %lu usecs [%f MB/s]\n", __func__, time_elapsed, (double)copy_tri_size / time_elapsed);

	gettimeofday(&start_time, NULL);

	cudaSafeCall(hipMemcpy(d_edges->edge_point0, edges.edge_point0, sizeof(point_t)*edge_count, hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->edge_point1, edges.edge_point1, sizeof(point_t)*edge_count, hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_edges->edge_mid_p, edges.edge_mid_p, sizeof(point_t)*edge_count, hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_edges->edge_len, edges.edge_len, sizeof(float)*edge_count, hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->edge_mark_tree_next, edges.edge_mark_tree_next, sizeof(int)*edge_count, hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->new_edge0, edges.new_edge0, sizeof(int)*edge_count, hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_edges->new_edge1, edges.new_edge1, sizeof(int)*edge_count, hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_edges->orth_edge, edges.orth_edge, sizeof(int)*edge_count, hipMemcpyHostToDevice));

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf("[%s] EDGES MEMCPY TIME ELAPSED = %lu usecs [%lf MB/s]\n", __func__, time_elapsed, (double)copy_edge_size / time_elapsed);

	return 0;
}

int malloc_copy_input_to_device2(void)
{
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	unsigned int edge_size, tri_size, copy_edge_size, copy_tri_size;

	edge_size = 3*sizeof(point_t)*edge_count + sizeof(float)*edge_count + 4*sizeof(int)*edge_count;
	tri_size = 8*sizeof(int)*tri_count;
	copy_edge_size = 3*sizeof(point_t)*(edge_count-new_edge_count) + 4*sizeof(int)*(edge_count-new_edge_count);
	copy_tri_size = 8*sizeof(int)*(tri_count-new_tri_count);

	printf("Total allocated memory = %u + %u bytes [%.1lf MB]\n", edge_size, tri_size, (double)(edge_size + tri_size)/(1024*1024));

	cudaSafeCall(hipMalloc((void**)&d_edges->edge_point0, sizeof(point_t)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->edge_point1, sizeof(point_t)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->edge_mid_p, sizeof(point_t)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->edge_len, sizeof(float)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->edge_mark_tree_next, sizeof(int)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->new_edge0, sizeof(int)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->new_edge1, sizeof(int)*edge_count));
	cudaSafeCall(hipMalloc((void**)&d_edges->orth_edge, sizeof(int)*edge_count));

	cudaSafeCall(hipMalloc((void**)&d_tris->tri_edge0, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_edge1, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_edge2, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_longest_edge, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_counter, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_counter_scan, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_rev_counter, sizeof(int)*tri_count));
	cudaSafeCall(hipMalloc((void**)&d_tris->tri_rev_counter_scan, sizeof(int)*tri_count));

	gettimeofday(&start_time, NULL);

	cudaSafeCall(hipMemcpy(d_tris->tri_edge0, tris.tri_edge0, sizeof(int)*(tri_count-new_tri_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_edge1, tris.tri_edge1, sizeof(int)*(tri_count-new_tri_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_edge2, tris.tri_edge2, sizeof(int)*(tri_count-new_tri_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_longest_edge, tris.tri_longest_edge, sizeof(int)*(tri_count-new_tri_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_counter, tris.tri_counter, sizeof(int)*(tri_count-new_tri_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_counter_scan, tris.tri_counter_scan, sizeof(int)*(tri_count-new_tri_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_rev_counter, tris.tri_rev_counter, sizeof(int)*(tri_count-new_tri_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_tris->tri_rev_counter_scan, tris.tri_rev_counter_scan, sizeof(int)*(tri_count-new_tri_count), hipMemcpyHostToDevice));

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf("[%s] TRIS MEMCPY TIME ELAPSED = %lu usecs [%f MB/s]\n", __func__, time_elapsed, (double)copy_tri_size / time_elapsed);

	gettimeofday(&start_time, NULL);

	cudaSafeCall(hipMemcpy(d_edges->edge_point0, edges.edge_point0, sizeof(point_t)*(edge_count-new_edge_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->edge_point1, edges.edge_point1, sizeof(point_t)*(edge_count-new_edge_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->edge_mid_p, edges.edge_mid_p, sizeof(point_t)*(edge_count-new_edge_count), hipMemcpyHostToDevice));
	//cudaSafeCall(hipMemcpy(d_edges->edge_len, edges.edge_len, sizeof(float)*(edge_count-new_edge_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->edge_mark_tree_next, edges.edge_mark_tree_next, sizeof(int)*(edge_count-new_edge_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->new_edge0, edges.new_edge0, sizeof(int)*(edge_count-new_edge_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->new_edge1, edges.new_edge1, sizeof(int)*(edge_count-new_edge_count), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_edges->orth_edge, edges.orth_edge, sizeof(int)*(edge_count-new_edge_count), hipMemcpyHostToDevice));

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf("[%s] EDGES MEMCPY TIME ELAPSED = %lu usecs [%lf MB/s]\n", __func__, time_elapsed, (double)copy_edge_size / time_elapsed);

	return 0;
}

char *readline(char *string, FILE *infile)
{
	char *result = NULL;
	
	/* Search for something that looks like a number. */
	do
	{
		result = fgets(string, MAX_LINE_SIZE, infile);
		if (result == (char *) NULL)
		{
			printf("Error:  Unexpected end of file\n");
			return NULL;
		}

		/* Skip anything that doesn't look like a number, a comment, */
		/*   or the end of a line.                                   */
		while ((*result != '\0') && (*result != '#')
			&& (*result != '.') && (*result != '+') && (*result != '-')
			&& ((*result < '0') || (*result > '9')))
		{
			result++;
		}
		/* If it's a comment or end of line, read another line and try again. */
	} while ((*result == '#') || (*result == '\0'));

	return result;
}

char *findfield(char *string)
{
	char *result;
	
	result = string;
	/* Skip the current field.  Stop upon reaching whitespace. */
	while ((*result != '\0') && (*result != '#')
		&& (*result != ' ') && (*result != '\t'))
	{
		result++;
	}
	/* Now skip the whitespace and anything else that doesn't look like a */
	/*   number, a comment, or the end of a line.                         */
	while ((*result != '\0') && (*result != '#')
		&& (*result != '.') && (*result != '+') && (*result != '-')
		&& ((*result < '0') || (*result > '9')))
	{
		result++;
	}
	/* Check for a comment (prefixed with `#'). */
	if (*result == '#')
	{
		*result = '\0';
	}

	return result;
}

int validate_input(void)
{
	int *bitmap;

	printf("Validating input... ");
	fflush(stdout);

#if 0
	// each point should belong to at least two edges
	bitmap = (int *)calloc(point_count, sizeof(int));
	FOR_EACH_E {
		bitmap[edge_arr[i].pp.points[0]]++;
		bitmap[edge_arr[i].pp.points[1]]++;
	}
	FOR_EACH_P {
		if (bitmap[i] < 2) {
			printf("failed for point[%d]\n", i);
			return -1;
		}
	}
	free(bitmap);
#endif
	// each edge should belong to at least one (positive) and at most two triangles
	bitmap = (int *)calloc(edge_count, sizeof(int));
	FOR_EACH_T {
		bitmap[abs(T_E0(i))] += (T_E0(i) >= 0 ? 1 : -1);
		bitmap[abs(T_E1(i))] += (T_E1(i) >= 0 ? 1 : -1);
		bitmap[abs(T_E2(i))] += (T_E2(i) >= 0 ? 1 : -1);

		if (T_LE(i) == 1) {
			refine_count++;
		}
	}

	FOR_EACH_E {
		if ((bitmap[i] < 0) || (bitmap[i] > 1)) {
			printf("failed for edge[%d]\n", i);
			return -1;
		}
	}
	free(bitmap);

	// check for zero or micro length edges
	FOR_EACH_E {
		if ((!COMPARE_FLOAT2(E_P0(i).x, E_P1(i).x)) && (!COMPARE_FLOAT2(E_P0(i).y, E_P1(i).y))) {
			printf("zero length edge[%d] detected!!!  [%.17g = %.17g] [%.17g = %.17g]\n", i, E_P0(i).x, E_P1(i).x, E_P0(i).y, E_P1(i).y);
			return -1;
		}
	}

	if (refine_count == 0) {
		printf(RED"WARNING! None of the triangles are choosed to refine? "NORM);
	} else {
		printf(BLUE"%d triangles to refine "NORM, refine_count);
	}

#if 0
	if (refine_count > 65000) {
		printf(RED"TOO MUCH TRIANGLES TO REFINE!!\n"NORM);
		exit(1);
	}

	if (refine_count > 60000) {
		printf(RED"WARNING: THIS MAY FAIL! "NORM);
	}
#endif
	printf("DONE!\n");
	return 0;
}

void print_input(void)
{
	printf("Points:\n");
	FOR_EACH_P {
		printf("point[%d]= %f %f\n", i, P(i).x, P(i).y);
	}

	printf("Edges:\n");
	FOR_EACH_E {
		printf("edge[%d]=[%f,%f]\t[%f,%f]\tmid=[%f,%f]\tlen=%f\tmark=%d\tnext=%d\tnew_edges={%d, %d, orth=%d}\n", i, E_P0(i).x, E_P0(i).y, E_P1(i).x, E_P1(i).y, E_MIDP(i).x, E_MIDP(i).y,  E_LEN(i), GET_E_MARK(i), (GET_E_NEXT(i) != NO_LINK) ? (int)GET_E_NEXT(i) : -1, E_NE0(i), E_NE1(i), E_OE(i));
	}

	printf("Triangles:\n");
	FOR_EACH_T {
		printf("triangle[%d]=%d %d %d longest=%d\tcounter=%d rev_counter=%d\tcounter_scan=%d counter_scan_rev=%d\n", i, T_E0(i), T_E1(i), T_E2(i), T_LE(i), GET_T_CNT(i), GET_T_RCNT(i), GET_T_CNT_SCAN(i), GET_T_RCNT_SCAN(i));
	}

	return;
}

int host_alloc_tris(void)
{
	tris.tri_edge0 = (int *)calloc(tri_count, sizeof(int));
	if (tris.tri_edge0 == NULL) {
		printf("cannot alloc tris.tri_edge0!\n");
		return -1;
	}

	tris.tri_edge1 = (int *)calloc(tri_count, sizeof(int));
	if (tris.tri_edge1 == NULL) {
		printf("cannot alloc tris.tri_edge1!\n");
		return -1;
	}

	tris.tri_edge2 = (int *)calloc(tri_count, sizeof(int));
	if (tris.tri_edge2 == NULL) {
		printf("cannot alloc tris.tri_edge2!\n");
		return -1;
	}

	tris.tri_longest_edge = (int *)calloc(tri_count, sizeof(int));
	if (tris.tri_longest_edge == NULL) {
		printf("cannot alloc tris.tri_longest_edge!\n");
		return -1;
	}

	tris.tri_counter = (unsigned int *)calloc(tri_count, sizeof(int));
	if (tris.tri_counter == NULL) {
		printf("cannot alloc tris.tri_counter!\n");
		return -1;
	}

	tris.tri_counter_scan = (unsigned int *)calloc(tri_count, sizeof(int));
	if (tris.tri_counter_scan == NULL) {
		printf("cannot alloc tris.tri_counter_scan!\n");
		return -1;
	}

	tris.tri_rev_counter = (unsigned int *)calloc(tri_count, sizeof(int));
	if (tris.tri_rev_counter == NULL) {
		printf("cannot alloc tris.tri_rev_counter!\n");
		return -1;
	}

	tris.tri_rev_counter_scan = (unsigned int *)calloc(tri_count, sizeof(int));
	if (tris.tri_rev_counter_scan == NULL) {
		printf("cannot alloc tris.tri_rev_counter_scan!\n");
		return -1;
	}

	return 0;
}

int read_triangles(FILE *infile)
{
	int count;
	char *lineptr;
	char line[MAX_LINE_SIZE];

	if ((lineptr = readline(line, infile)) == NULL) {
		printf("error on reading line!\n");
		return -1;
	}

	// get the triangle count
	sscanf(lineptr, "%d", &count);
	printf("%d triangles, ", count);
	fflush(stdout);

	if (count <= 0) {
		printf("invalid # of triangle\n");
		return -1;
	}

	tri_count = count;

	if (host_alloc_tris() < 0) {
		printf("host alloc error\n");
		return -1;
	}

	for (int i = 0; i < count; i++) {
		if ((lineptr = readline(line, infile)) == NULL) {
			printf("error on reading line!\n");
			return -1;
		}
		sscanf(lineptr, "%d %d %d %d", &T_E0(i), &T_E1(i), &T_E2(i), &T_LE(i));
	}

	return 0;
}

int host_alloc_edges(void)
{
	edges.edge_point0 = (point_t *)calloc(edge_count, sizeof(point_t));
	if (edges.edge_point0 == NULL) {
		printf("cannot alloc edge_point0 arr!\n");
		return -1;
	}

	edges.edge_point1 = (point_t *)calloc(edge_count, sizeof(point_t));
	if (edges.edge_point1 == NULL) {
		printf("cannot alloc edge_point1 arr!\n");
		return -1;
	}

	edges.edge_mid_p = (point_t *)calloc(edge_count, sizeof(point_t));
	if (edges.edge_mid_p == NULL) {
		printf("cannot alloc edge_mid_p arr!\n");
		return -1;
	}

	edges.edge_len = (float *)calloc(edge_count, sizeof(float));
	if (edges.edge_len == NULL) {
		printf("cannot alloc edge_len arr!\n");
		return -1;
	}

	edges.edge_mark_tree_next = (unsigned int *)calloc(edge_count, sizeof(int));
	if (edges.edge_mark_tree_next == NULL) {
		printf("cannot alloc edge_mark_tree_next arr!\n");
		return -1;
	}

	edges.new_edge0 = (int *)calloc(edge_count, sizeof(int));
	if (edges.new_edge0 == NULL) {
		printf("cannot alloc new_edge0 arr!\n");
		return -1;
	}

	edges.new_edge1 = (int *)calloc(edge_count, sizeof(int));
	if (edges.new_edge1 == NULL) {
		printf("cannot alloc new_edge1 arr!\n");
		return -1;
	}

	edges.orth_edge = (int *)calloc(edge_count, sizeof(int));
	if (edges.orth_edge == NULL) {
		printf("cannot alloc orth_edge arr!\n");
		return -1;
	}

	return 0;
}

int read_edges(FILE *infile)
{
	int count;
	char *lineptr;
	char line[MAX_LINE_SIZE];
	int p[2];

	if ((lineptr = readline(line, infile)) == NULL) {
		printf("error on reading line!\n");
		return -1;
	}

	// get the edge count
	sscanf(lineptr, "%d", &count);
	printf("%d edges, ", count);
	fflush(stdout);

	if (count <= 0) {
		printf("invalid # of edges\n");
		return -1;
	}

	edge_count = count + 1;	// 0th edge is invalid

	if (host_alloc_edges() < 0) {
		printf("host alloc error\n");
		return -1;
	}

	for (int i = 0; i < count; i++) {
		if ((lineptr = readline(line, infile)) == NULL) {
			printf("error on reading line!\n");
			return -1;
		}
		sscanf(lineptr, "%d %d", &p[0], &p[1]);
		E_P0(i+1) = P(p[0]);
		E_P1(i+1) = P(p[1]);
	}

	return 0;
}

int read_points(FILE *infile)
{
	int count;
	point_t point = {0};
	char *lineptr;
	char line[MAX_LINE_SIZE];

	if ((lineptr = readline(line, infile)) == NULL) {
		printf("error on reading line!\n");
		return -1;
	}

	// get the point count
	sscanf(lineptr, "%d", &count);
	printf("%d points, ", count);
	fflush(stdout);

	if (count <= 0) {
		printf("invalid # of points\n");
		return -1;
	}

	point_arr = (point_t *)calloc(count, sizeof(point_t));
	if (point_arr == NULL) {
		printf("cannot alloc point_arr!\n");
		return -1;
	}

	for (int i = 0; i < count; i++) {
		if ((lineptr = readline(line, infile)) == NULL) {
			printf("error on reading line!\n");
			return -1;
		}
		sscanf(lineptr, "%f %f", &point.x, &point.y);
		P(i) = point;
	}

	point_count = count;

	return 0;
}

int read_input_file(FILE *infile)
{
	printf("Reading input file... ");
	fflush(stdout);

	if (read_points(infile) < 0) {
		printf("error on reading points!\n");
		return -1;
	}

	if (read_edges(infile) < 0) {
		printf("error on reading edges!\n");
		return -1;
	}

	if (read_triangles(infile) < 0) {
		printf("error on reading triangles!\n");
		return -1;
	}

	printf("DONE!\n");
	return 0;
}

int write_edges(FILE *outfile)
{
	printf("Writing %d edges... ", edge_count - 1);
	fflush(stdout);

	fprintf(outfile, "#edges\n");
	fprintf(outfile, "%d\n", edge_count - 1);

	FOR_EACH_E {
		fprintf(outfile, "[%.17g %.17g] \t[%.17g %.17g]\n", E_P0(i).x, E_P0(i).y, E_P1(i).x, E_P1(i).y);
	}

	fprintf(outfile, "\n");

	printf("OK, ");
	return 0;
}

int write_triangles(FILE *outfile)
{
	printf("Writing %d triangles... ", tri_count);
	fflush(stdout);

	fprintf(outfile, "#triangles\n");
	fprintf(outfile, "%d\n", tri_count);

	FOR_EACH_T {
		// TODO: refine edilenleri bul
		fprintf(outfile, "%d \t%d \t%d \t%d\n", T_E0(i), T_E1(i), T_E2(i), 0);
	}

	fprintf(outfile, "\n");

	printf("OK, ");
	return 0;
}

int write_output(FILE *outfile)
{
	printf("Generating output file... ");
	fflush(stdout);

#if 0
	if (write_points(outfile) < 0) {
		printf("error on writing points!\n");
		return -1;
	}
#endif
	if (write_edges(outfile) < 0) {
		printf("error on writing edges!\n");
		return -1;
	}

	if (write_triangles(outfile) < 0) {
		printf("error on writing triangles!\n");
		return -1;
	}

	printf("DONE!\n");
	return 0;
}

int read_node_points(FILE *infile)
{
	int count;
	point_t point = {0};
	char *lineptr;
	char line[MAX_LINE_SIZE];

	if ((lineptr = readline(line, infile)) == NULL) {
		printf("error on reading line!\n");
		return -1;
	}

	// get the point count
	sscanf(lineptr, "%d", &count);
	printf("%d points, ", count);
	fflush(stdout);

	if (count <= 0) {
		printf("invalid # of points\n");
		return -1;
	}

	point_arr = (point_t *)calloc(count, sizeof(point_t));
	if (point_arr == NULL) {
		printf("cannot alloc point_arr!\n");
		return -1;
	}

	for (int i = 0; i < count; i++) {
		if ((lineptr = readline(line, infile)) == NULL) {
			printf("error on reading line!\n");
			return -1;
		}
		strtol(lineptr, &lineptr, 0);
		lineptr = findfield(lineptr);
		point.x = strtof(lineptr, &lineptr);
		lineptr = findfield(lineptr);
		point.y = strtof(lineptr, &lineptr);

		P(i) = point;
	}

	point_count = count;

	return 0;
}

int read_node_file(FILE *infile)
{
	printf("Reading node file... ");
	fflush(stdout);

	if (read_node_points(infile) < 0) {
		printf("error on reading points!\n");
		return -1;
	}

	printf("DONE!\n");
	return 0;
}

#define HASH_TABLE_SIZE		(point_count+1)

int *hash_table;

int hash(int i)
{
	return i % HASH_TABLE_SIZE;
}

// check if two edges are equal and return 1 if they are equal. else, return 0.
int edge_equals(in_edge_t *e1, in_edge_t *e2)
{
	if ((e1->point_ix[0] == e2->point_ix[0]) && (e1->point_ix[1] == e2->point_ix[1])) {
		return 1;
	}

	return 0;
}

// return index if edge is found
// return 0 if edge is not found
int find_edge(int p1, int p2)
{
	in_edge_t e;
	int h = hash(p1);
	int index = hash_table[h];

	e.point_ix[0] = p1;
	e.point_ix[1] = p2;

	while (index != 0) {
		if (edge_equals(&e, &in_edge_arr[index])) {
			//printf("found edge [%d %d] in %d\n", p1, p2, index);
			break;
		}
		index = in_edge_arr[index].next;
	}

	return index;
}

int create_edge(int p1, int p2)
{
	in_edge_t e;
	int h;

	e.point_ix[0] = p1;
	e.point_ix[1] = p2;

	in_edge_count++;

	// add to hash table
	h = hash(p1);
	//printf("%d: creating [%d %d] hash=%d\n", in_edge_count, p1, p2, h);
	e.next = hash_table[h];
	hash_table[h] = in_edge_count;

	in_edge_arr[in_edge_count] = e;

	return in_edge_count;
}

int create_edges_and_triangles(void)
{
	//int r;

	printf("Creating edges... ");
	fflush(stdout);

	// alloc tri arr
	if (host_alloc_tris() < 0) {
		printf("host alloc error\n");
		return -1;
	}

	// alloc max # of possible edges, i.e. 3xtri_count
	in_edge_count = tri_count * 3;
	in_edge_arr = (in_edge_t *)calloc(in_edge_count, sizeof(in_edge_t));
	if (in_edge_arr == NULL) {
		printf("cannot alloc in_edge_arr!\n");
		return -1;
	}

	// alloc hash table
	hash_table = (int *)calloc(HASH_TABLE_SIZE, sizeof(int));
	if (hash_table == NULL) {
		printf("cannot alloc hash_table!\n");
		return -1;
	}

	in_edge_count = 0;

	//r = get_rand_int(0, tri_count);

	for (int i = 0; i < tri_count; i++) {
		for (int j = 0; j < 3; j++) {
			int p1 = tri_point_arr[i].point[j];
			int p2 = tri_point_arr[i].point[(j + 1) % 3];
			int index = -find_edge(p2, p1);	// search the reverse of the edge

			if (!index)
				index = create_edge(p1, p2);

			if (j == 0)
				T_E0(i) = index;
			else if (j == 1)
				T_E1(i) = index;
			else
				T_E2(i) = index;
		}

		//if (get_rand_int(0, tri_count) < (r/100 + 1))
		if (get_rand_int(0, 1000) < 1)
			T_LE(i) = 1;
	}

	printf("%d edges, ", in_edge_count);

	edge_count = in_edge_count + 1;

	if (host_alloc_edges() < 0) {
		printf("host alloc error\n");
		return -1;
	}

	for (int i = 1; i <= in_edge_count; i++) {
		int p[2];

		p[0] = in_edge_arr[i].point_ix[0];
		p[1] = in_edge_arr[i].point_ix[1];

		E_P0(i) = P(p[0]-1);
		E_P1(i) = P(p[1]-1);
	}

	return 0;
}

int read_tri_points(FILE *infile)
{
	int count;
	tri_point_t tp;
	char *lineptr;
	char line[MAX_LINE_SIZE];

	if ((lineptr = readline(line, infile)) == NULL) {
		printf("error on reading line!\n");
		return -1;
	}

	// get the tri count
	sscanf(lineptr, "%d", &count);
	printf("%d triangles, ", count);
	fflush(stdout);

	if (count <= 0) {
		printf("invalid # of triangles\n");
		return -1;
	}

	tri_point_arr = (tri_point_t *)calloc(count, sizeof(tri_point_t));
	if (tri_point_arr == NULL) {
		printf("cannot alloc tri_point_arr!\n");
		return -1;
	}

	for (int i = 0; i < count; i++) {
		if ((lineptr = readline(line, infile)) == NULL) {
			printf("error on reading line!\n");
			return -1;
		}
		strtol(lineptr, &lineptr, 0);

		for (int j = 0; j < 3; j++) {
			lineptr = findfield(lineptr);
			tp.point[j] = (int)strtod(lineptr, &lineptr);
		}

		tri_point_arr[i] = tp;
	}

	tri_count = count;

	return 0;
}

int read_ele_file(FILE *infile)
{
	printf("Reading ele file... ");
	fflush(stdout);

	srand(time(NULL));

	if (read_tri_points(infile) < 0) {
		printf("error on reading tri_points!\n");
		return -1;
	}

	if (create_edges_and_triangles() < 0) {
		printf("error on create_edges_and_triangles!\n");
		return -1;
	}

	free(tri_point_arr);
	tri_point_arr = NULL;

	free(in_edge_arr);
	in_edge_arr = NULL;

	free(hash_table);
	hash_table = NULL;

	printf("DONE!\n");
	return 0;
}

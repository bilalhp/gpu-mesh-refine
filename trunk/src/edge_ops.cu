#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <sys/time.h>

#include <main.h>

#if 0
static inline __device__ point_t calc_mid_point(point_t *p1, point_t *p2)
{
	point_t ret;

	ret.x = (p1->x + p2->x) / 2;
	ret.y = (p1->y + p2->y) / 2;

	return ret;
}
#endif

__global__ void cu_follow_links(unsigned int *edge_mark_tree_next, int edge_count)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int mn, next_mn;
	unsigned int next_ix;

	if ((tid == 0) || (tid >= edge_count))
		return;

	mn = edge_mark_tree_next[tid];
	if (!(__GET_E_MARK(mn) & 2)) {	// not marked
		return;
	}

	next_ix = __GET_E_NEXT(mn);
	if (!next_ix || (next_ix == NO_LINK)) {
		return;
	}

	while (next_ix && (next_ix != NO_LINK)) {
		next_mn = edge_mark_tree_next[next_ix];
		if (__GET_E_MARK(next_mn) & 2) {	// next is already marked
			break;
		}
		__SET_E_NEXT(mn, __GET_E_NEXT(next_mn));
		next_mn |= 2;
		__SET_E_NEXT(next_mn, NO_LINK);
		edge_mark_tree_next[next_ix] = next_mn;
		next_ix = __GET_E_NEXT(mn);
	}

	__SET_E_NEXT(mn, NO_LINK);
	edge_mark_tree_next[tid] = mn;
}

__global__ void cu_correct_marks(float *edge_len, unsigned int *edge_mark_tree_next, int edge_count)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;

	if ((tid == 0) || (tid >= edge_count))
		return;

	if (edge_len[tid] < 0) {
		edge_mark_tree_next[tid] |= 2;
	}
}

__global__ void cu_establish_links(int *tri_edge0, int *tri_edge1, int *tri_edge2, int *tri_longest_edge, unsigned int *edge_mark_tree_next, int tri_count)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	int edge, ledge;
	unsigned int mn;

	if (tid >= tri_count)
		return;

	__syncthreads();
	ledge = tri_longest_edge[tid];

	edge = abs(tri_edge0[tid]);
	if (edge != ledge) {
		mn = edge_mark_tree_next[edge];
		if (__GET_E_MARK(mn) & 1) {
			__SET_E_NEXT(mn, ledge);
			edge_mark_tree_next[edge] = mn;
		}
	}

	__syncthreads();
	edge = abs(tri_edge1[tid]);
	if (edge != ledge) {
		mn = edge_mark_tree_next[edge];
		if (__GET_E_MARK(mn) & 1) {
			__SET_E_NEXT(mn, ledge);
			edge_mark_tree_next[edge] = mn;
		}
	}

	__syncthreads();
	edge = abs(tri_edge2[tid]);
	if (edge != ledge) {
		mn = edge_mark_tree_next[edge];
		if (__GET_E_MARK(mn) & 1) {
			__SET_E_NEXT(mn, ledge);
			edge_mark_tree_next[edge] = mn;
		}
	}
}

__global__ void cu_mark_longest(int *tri_edge0, int *tri_edge1, int *tri_edge2, int *tri_longest_edge, float *edge_len, unsigned int *edge_mark_tree_next, int tri_count)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	float llen = 0, clen;
	int ledge;
	int edge_ix;

	if (tid >= tri_count)
		return;

	__syncthreads();
	edge_ix = abs(tri_edge0[tid]);
	clen = abs(edge_len[edge_ix]);
	if (clen > llen) {
		llen = clen;
		ledge = edge_ix;
	}

	__syncthreads();
	edge_ix = abs(tri_edge1[tid]);
	clen = abs(edge_len[edge_ix]);
	if (clen > llen) {
		llen = clen;
		ledge = edge_ix;
	}

	__syncthreads();
	edge_ix = abs(tri_edge2[tid]);
	clen = abs(edge_len[edge_ix]);
	if (clen > llen) {
		llen = clen;
		ledge = edge_ix;
	}

	if (tri_longest_edge[tid] == 1) {
		edge_len[ledge] = -llen;	// negative edge len is blackmark
	}

	edge_mark_tree_next[ledge] = 1;		// mark as longest edge
	__syncthreads();
	tri_longest_edge[tid] = ledge;
}

__global__ void cu_calc_edge_len_mid_p(point_t *edge_point0, point_t *edge_point1, point_t *edge_mid_p, float *edge_len, int edge_count)
{
	unsigned int tid = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	point_t p0, p1;

	if ((tid == 0) || (tid >= edge_count))
		return;

	//__syncthreads();
	p0 = edge_point0[tid];
	p1 = edge_point1[tid];

	//__syncthreads();
	edge_len[tid] = sqrtf(powf(fabs(p0.x - p1.x), 2) + powf(fabs(p0.y - p1.y), 2));
	//__syncthreads();
	edge_mid_p[tid].x = (p0.x + p1.x) / 2;
	edge_mid_p[tid].y = (p0.y + p1.y) / 2;
}

void follow_links(void)
{
	hipError_t err;
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	int block_count = edge_count/threadsPerBlock + 1;
	dim3 dimGrid(65535, block_count/65535 + 1);

	printf(DGREEN"[%s]"NORM" WORK STARTED: #threads=%d #blocks=%d***\n", __func__, threadsPerBlock, block_count);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

#if (__CUDA_ARCH__ >= 200)
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_correct_marks), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_follow_links), hipFuncCachePreferL1);
#endif
	cu_correct_marks<<<dimGrid, threadsPerBlock>>>(d_edges->edge_len, d_edges->edge_mark_tree_next, edge_count);
	cu_follow_links<<<dimGrid, threadsPerBlock>>>(d_edges->edge_mark_tree_next, edge_count);

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("error!\n");
	}
}

void establish_links(void)
{
	hipError_t err;
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	int block_count = tri_count/threadsPerBlock + 1;
	dim3 dimGrid(65535, block_count/65535 + 1);

	printf(DGREEN"[%s]"NORM" WORK STARTED: #threads=%d #blocks=%d***\n", __func__, threadsPerBlock, block_count);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

#if (__CUDA_ARCH__ >= 200)
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_establish_links), hipFuncCachePreferL1);
#endif
	cu_establish_links<<<dimGrid, threadsPerBlock>>>(d_tris->tri_edge0, d_tris->tri_edge1, d_tris->tri_edge2, d_tris->tri_longest_edge, d_edges->edge_mark_tree_next, tri_count);

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("error!\n");
	}
}

void mark_longest_edges(void)
{
	hipError_t err;
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	int block_count = tri_count/threadsPerBlock + 1;
	dim3 dimGrid(65535, block_count/65535 + 1);

	printf(DGREEN"[%s]"NORM" WORK STARTED: #threads=%d #blocks=%d***\n", __func__, threadsPerBlock, block_count);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

#if (__CUDA_ARCH__ >= 200)
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_mark_longest), hipFuncCachePreferL1);
#endif
	cu_mark_longest<<<dimGrid, threadsPerBlock>>>(d_tris->tri_edge0, d_tris->tri_edge1, d_tris->tri_edge2, d_tris->tri_longest_edge, d_edges->edge_len, d_edges->edge_mark_tree_next, tri_count);

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("error!\n");
	}
}

void calc_edge_lengths_mid_p(void)
{
	hipError_t err;
	struct timeval start_time, end_time;
	unsigned long time_elapsed = 0;
	int block_count = edge_count/threadsPerBlock + 1;
	dim3 dimGrid(65535, block_count/65535 + 1);

	printf(DGREEN"[%s]"NORM" WORK STARTED: #threads=%d #blocks=%d***\n", __func__, threadsPerBlock, block_count);
	fflush(stdout);
	usleep(100000);
	gettimeofday(&start_time, NULL);

#if (__CUDA_ARCH__ >= 200)
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(cu_calc_edge_len_mid_p), hipFuncCachePreferL1);
#endif
	cu_calc_edge_len_mid_p<<<dimGrid, threadsPerBlock>>>(d_edges->edge_point0, d_edges->edge_point1, d_edges->edge_mid_p, d_edges->edge_len, edge_count);

	cudaSafeCall(hipDeviceSynchronize());

	gettimeofday(&end_time, NULL);
	time_elapsed = get_time_diff_us(&start_time, &end_time);

	printf(DGREEN"[%s]"NORM" TIME ELAPSED = %lu usecs\n", __func__, time_elapsed);

	// check err
	err = hipGetLastError();
	if (hipSuccess != err) {
		printf("error!\n");
	}
}
